#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <fstream>
#include <time.h>

#define CLEANUP(s)                                   \
do {                                                 \
    printf ("%s\n", s);                              \
    if (yHostPtr)           free(yHostPtr);          \
    if (zHostPtr)           free(zHostPtr);          \
    if (xIndHostPtr)        free(xIndHostPtr);       \
    if (xValHostPtr)        free(xValHostPtr);       \
    if (cooRowIndexHostPtr) free(cooRowIndexHostPtr);\
    if (cooColIndexHostPtr) free(cooColIndexHostPtr);\
    if (cooValHostPtr)      free(cooValHostPtr);     \
    if (y)                  hipFree(y);             \
    if (z)                  hipFree(z);             \
    if (xInd)               hipFree(xInd);          \
    if (xVal)               hipFree(xVal);          \
    if (csrRowPtr)          hipFree(csrRowPtr);     \
    if (cooRowIndex)        hipFree(cooRowIndex);   \
    if (cooColIndex)        hipFree(cooColIndex);   \
    if (cooVal)             hipFree(cooVal);        \
    if (descr)              hipsparseDestroyMatDescr(descr);\
    if (handle)             hipsparseDestroy(handle); \
    hipDeviceReset();          \
    fflush (stdout);                                 \
} while (0)

int main(){
    hipError_t cudaStat1,cudaStat2,cudaStat3,cudaStat4;
    hipsparseStatus_t status;
    hipsparseHandle_t handle=0;
    hipsparseMatDescr_t descr=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int *    cooRowIndexHostPtr=0;
    int *    cooColIndexHostPtr=0;
    double * cooValHostPtr=0;
    int *    cooRowIndex=0;
    int *    cooColIndex=0;
    double * cooVal=0;
    int *    xIndHostPtr=0;
    double * xValHostPtr=0;
    double * yHostPtr=0;
    double * y_static=0;
    int *    xInd=0;
    double * xVal=0;
    double * y=0;
    int *    csrRowPtr=0;
    double * zHostPtr=0;
    double * z=0;
    int      n, nnz;
    double dzero =0.0;
    double done = 1.0;

    printf("testing example\n");
    /* create the following sparse test matrix in COO format */
    /* |1.0     2.0 3.0     |
       |    4.0             |
       |5.0     6.0 7.0     |
       |    8.0     9.0     |
       |                10.0| */

    n = 200;      // rank of the matrix
    nnz = 796;   // number of non-zero elements
    
    cooRowIndexHostPtr = (int *)   malloc(nnz*sizeof(cooRowIndexHostPtr[0]));
    cooColIndexHostPtr = (int *)   malloc(nnz*sizeof(cooColIndexHostPtr[0]));
    cooValHostPtr      = (double *)malloc(nnz*sizeof(cooValHostPtr[0]));
    if ((!cooRowIndexHostPtr) || (!cooColIndexHostPtr) || (!cooValHostPtr)){
        CLEANUP("Host malloc failed (matrix)");
        return 1;
    }

    cooRowIndexHostPtr[0]=0; cooColIndexHostPtr[0]=0; cooValHostPtr[0]=-615.6962723589505;
    cooRowIndexHostPtr[1]=0; cooColIndexHostPtr[1]=1; cooValHostPtr[1]=310.0731361794753;
    cooRowIndexHostPtr[2]=0; cooColIndexHostPtr[2]=100; cooValHostPtr[2]=4.0;
    cooRowIndexHostPtr[3]=1; cooColIndexHostPtr[3]=0; cooValHostPtr[3]=310.0731361794753;
    cooRowIndexHostPtr[4]=1; cooColIndexHostPtr[4]=1; cooValHostPtr[4]=-615.6962723589505;
    cooRowIndexHostPtr[5]=1; cooColIndexHostPtr[5]=2; cooValHostPtr[5]=310.0731361794753;
    cooRowIndexHostPtr[6]=1; cooColIndexHostPtr[6]=101; cooValHostPtr[6]=4.0;
    cooRowIndexHostPtr[7]=2; cooColIndexHostPtr[7]=1; cooValHostPtr[7]=310.0731361794753;
    cooRowIndexHostPtr[8]=2; cooColIndexHostPtr[8]=2; cooValHostPtr[8]=-615.6962723589505;
    cooRowIndexHostPtr[9]=2; cooColIndexHostPtr[9]=3; cooValHostPtr[9]=310.0731361794753;
    cooRowIndexHostPtr[10]=2; cooColIndexHostPtr[10]=102; cooValHostPtr[10]=4.0;
    cooRowIndexHostPtr[11]=3; cooColIndexHostPtr[11]=2; cooValHostPtr[11]=310.0731361794753;
    cooRowIndexHostPtr[12]=3; cooColIndexHostPtr[12]=3; cooValHostPtr[12]=-615.6962723589505;
    cooRowIndexHostPtr[13]=3; cooColIndexHostPtr[13]=4; cooValHostPtr[13]=310.0731361794753;
    cooRowIndexHostPtr[14]=3; cooColIndexHostPtr[14]=103; cooValHostPtr[14]=4.0;
    cooRowIndexHostPtr[15]=4; cooColIndexHostPtr[15]=3; cooValHostPtr[15]=310.0731361794753;
    cooRowIndexHostPtr[16]=4; cooColIndexHostPtr[16]=4; cooValHostPtr[16]=-615.6962723589505;
    cooRowIndexHostPtr[17]=4; cooColIndexHostPtr[17]=5; cooValHostPtr[17]=310.0731361794753;
    cooRowIndexHostPtr[18]=4; cooColIndexHostPtr[18]=104; cooValHostPtr[18]=4.0;
    cooRowIndexHostPtr[19]=5; cooColIndexHostPtr[19]=4; cooValHostPtr[19]=310.0731361794753;
    cooRowIndexHostPtr[20]=5; cooColIndexHostPtr[20]=5; cooValHostPtr[20]=-615.6962723589505;
    cooRowIndexHostPtr[21]=5; cooColIndexHostPtr[21]=6; cooValHostPtr[21]=310.0731361794753;
    cooRowIndexHostPtr[22]=5; cooColIndexHostPtr[22]=105; cooValHostPtr[22]=4.0;
    cooRowIndexHostPtr[23]=6; cooColIndexHostPtr[23]=5; cooValHostPtr[23]=310.0731361794753;
    cooRowIndexHostPtr[24]=6; cooColIndexHostPtr[24]=6; cooValHostPtr[24]=-615.6962723589505;
    cooRowIndexHostPtr[25]=6; cooColIndexHostPtr[25]=7; cooValHostPtr[25]=310.0731361794753;
    cooRowIndexHostPtr[26]=6; cooColIndexHostPtr[26]=106; cooValHostPtr[26]=4.0;
    cooRowIndexHostPtr[27]=7; cooColIndexHostPtr[27]=6; cooValHostPtr[27]=310.0731361794753;
    cooRowIndexHostPtr[28]=7; cooColIndexHostPtr[28]=7; cooValHostPtr[28]=-615.6962723589505;
    cooRowIndexHostPtr[29]=7; cooColIndexHostPtr[29]=8; cooValHostPtr[29]=310.0731361794753;
    cooRowIndexHostPtr[30]=7; cooColIndexHostPtr[30]=107; cooValHostPtr[30]=4.0;
    cooRowIndexHostPtr[31]=8; cooColIndexHostPtr[31]=7; cooValHostPtr[31]=310.0731361794753;
    cooRowIndexHostPtr[32]=8; cooColIndexHostPtr[32]=8; cooValHostPtr[32]=-615.6962723589505;
    cooRowIndexHostPtr[33]=8; cooColIndexHostPtr[33]=9; cooValHostPtr[33]=310.0731361794753;
    cooRowIndexHostPtr[34]=8; cooColIndexHostPtr[34]=108; cooValHostPtr[34]=4.0;
    cooRowIndexHostPtr[35]=9; cooColIndexHostPtr[35]=8; cooValHostPtr[35]=310.0731361794753;
    cooRowIndexHostPtr[36]=9; cooColIndexHostPtr[36]=9; cooValHostPtr[36]=-615.6962723589505;
    cooRowIndexHostPtr[37]=9; cooColIndexHostPtr[37]=10; cooValHostPtr[37]=310.0731361794753;
    cooRowIndexHostPtr[38]=9; cooColIndexHostPtr[38]=109; cooValHostPtr[38]=4.0;
    cooRowIndexHostPtr[39]=10; cooColIndexHostPtr[39]=9; cooValHostPtr[39]=310.0731361794753;
    cooRowIndexHostPtr[40]=10; cooColIndexHostPtr[40]=10; cooValHostPtr[40]=-615.6962723589505;
    cooRowIndexHostPtr[41]=10; cooColIndexHostPtr[41]=11; cooValHostPtr[41]=310.0731361794753;
    cooRowIndexHostPtr[42]=10; cooColIndexHostPtr[42]=110; cooValHostPtr[42]=4.0;
    cooRowIndexHostPtr[43]=11; cooColIndexHostPtr[43]=10; cooValHostPtr[43]=310.0731361794753;
    cooRowIndexHostPtr[44]=11; cooColIndexHostPtr[44]=11; cooValHostPtr[44]=-615.6962723589505;
    cooRowIndexHostPtr[45]=11; cooColIndexHostPtr[45]=12; cooValHostPtr[45]=310.0731361794753;
    cooRowIndexHostPtr[46]=11; cooColIndexHostPtr[46]=111; cooValHostPtr[46]=4.0;
    cooRowIndexHostPtr[47]=12; cooColIndexHostPtr[47]=11; cooValHostPtr[47]=310.0731361794753;
    cooRowIndexHostPtr[48]=12; cooColIndexHostPtr[48]=12; cooValHostPtr[48]=-615.6962723589505;
    cooRowIndexHostPtr[49]=12; cooColIndexHostPtr[49]=13; cooValHostPtr[49]=310.0731361794753;
    cooRowIndexHostPtr[50]=12; cooColIndexHostPtr[50]=112; cooValHostPtr[50]=4.0;
    cooRowIndexHostPtr[51]=13; cooColIndexHostPtr[51]=12; cooValHostPtr[51]=310.0731361794753;
    cooRowIndexHostPtr[52]=13; cooColIndexHostPtr[52]=13; cooValHostPtr[52]=-615.6962723589505;
    cooRowIndexHostPtr[53]=13; cooColIndexHostPtr[53]=14; cooValHostPtr[53]=310.0731361794753;
    cooRowIndexHostPtr[54]=13; cooColIndexHostPtr[54]=113; cooValHostPtr[54]=4.0;
    cooRowIndexHostPtr[55]=14; cooColIndexHostPtr[55]=13; cooValHostPtr[55]=310.0731361794753;
    cooRowIndexHostPtr[56]=14; cooColIndexHostPtr[56]=14; cooValHostPtr[56]=-615.6962723589505;
    cooRowIndexHostPtr[57]=14; cooColIndexHostPtr[57]=15; cooValHostPtr[57]=310.0731361794753;
    cooRowIndexHostPtr[58]=14; cooColIndexHostPtr[58]=114; cooValHostPtr[58]=4.0;
    cooRowIndexHostPtr[59]=15; cooColIndexHostPtr[59]=14; cooValHostPtr[59]=310.0731361794753;
    cooRowIndexHostPtr[60]=15; cooColIndexHostPtr[60]=15; cooValHostPtr[60]=-615.6962723589505;
    cooRowIndexHostPtr[61]=15; cooColIndexHostPtr[61]=16; cooValHostPtr[61]=310.0731361794753;
    cooRowIndexHostPtr[62]=15; cooColIndexHostPtr[62]=115; cooValHostPtr[62]=4.0;
    cooRowIndexHostPtr[63]=16; cooColIndexHostPtr[63]=15; cooValHostPtr[63]=310.0731361794753;
    cooRowIndexHostPtr[64]=16; cooColIndexHostPtr[64]=16; cooValHostPtr[64]=-615.6962723589505;
    cooRowIndexHostPtr[65]=16; cooColIndexHostPtr[65]=17; cooValHostPtr[65]=310.0731361794753;
    cooRowIndexHostPtr[66]=16; cooColIndexHostPtr[66]=116; cooValHostPtr[66]=4.0;
    cooRowIndexHostPtr[67]=17; cooColIndexHostPtr[67]=16; cooValHostPtr[67]=310.0731361794753;
    cooRowIndexHostPtr[68]=17; cooColIndexHostPtr[68]=17; cooValHostPtr[68]=-615.6962723589505;
    cooRowIndexHostPtr[69]=17; cooColIndexHostPtr[69]=18; cooValHostPtr[69]=310.0731361794753;
    cooRowIndexHostPtr[70]=17; cooColIndexHostPtr[70]=117; cooValHostPtr[70]=4.0;
    cooRowIndexHostPtr[71]=18; cooColIndexHostPtr[71]=17; cooValHostPtr[71]=310.0731361794753;
    cooRowIndexHostPtr[72]=18; cooColIndexHostPtr[72]=18; cooValHostPtr[72]=-615.6962723589505;
    cooRowIndexHostPtr[73]=18; cooColIndexHostPtr[73]=19; cooValHostPtr[73]=310.0731361794753;
    cooRowIndexHostPtr[74]=18; cooColIndexHostPtr[74]=118; cooValHostPtr[74]=4.0;
    cooRowIndexHostPtr[75]=19; cooColIndexHostPtr[75]=18; cooValHostPtr[75]=310.0731361794753;
    cooRowIndexHostPtr[76]=19; cooColIndexHostPtr[76]=19; cooValHostPtr[76]=-615.6962723589505;
    cooRowIndexHostPtr[77]=19; cooColIndexHostPtr[77]=20; cooValHostPtr[77]=310.0731361794753;
    cooRowIndexHostPtr[78]=19; cooColIndexHostPtr[78]=119; cooValHostPtr[78]=4.0;
    cooRowIndexHostPtr[79]=20; cooColIndexHostPtr[79]=19; cooValHostPtr[79]=310.0731361794753;
    cooRowIndexHostPtr[80]=20; cooColIndexHostPtr[80]=20; cooValHostPtr[80]=-615.6962723589505;
    cooRowIndexHostPtr[81]=20; cooColIndexHostPtr[81]=21; cooValHostPtr[81]=310.0731361794753;
    cooRowIndexHostPtr[82]=20; cooColIndexHostPtr[82]=120; cooValHostPtr[82]=4.0;
    cooRowIndexHostPtr[83]=21; cooColIndexHostPtr[83]=20; cooValHostPtr[83]=310.0731361794753;
    cooRowIndexHostPtr[84]=21; cooColIndexHostPtr[84]=21; cooValHostPtr[84]=-615.6962723589505;
    cooRowIndexHostPtr[85]=21; cooColIndexHostPtr[85]=22; cooValHostPtr[85]=310.0731361794753;
    cooRowIndexHostPtr[86]=21; cooColIndexHostPtr[86]=121; cooValHostPtr[86]=4.0;
    cooRowIndexHostPtr[87]=22; cooColIndexHostPtr[87]=21; cooValHostPtr[87]=310.0731361794753;
    cooRowIndexHostPtr[88]=22; cooColIndexHostPtr[88]=22; cooValHostPtr[88]=-615.6962723589505;
    cooRowIndexHostPtr[89]=22; cooColIndexHostPtr[89]=23; cooValHostPtr[89]=310.0731361794753;
    cooRowIndexHostPtr[90]=22; cooColIndexHostPtr[90]=122; cooValHostPtr[90]=4.0;
    cooRowIndexHostPtr[91]=23; cooColIndexHostPtr[91]=22; cooValHostPtr[91]=310.0731361794753;
    cooRowIndexHostPtr[92]=23; cooColIndexHostPtr[92]=23; cooValHostPtr[92]=-615.6962723589505;
    cooRowIndexHostPtr[93]=23; cooColIndexHostPtr[93]=24; cooValHostPtr[93]=310.0731361794753;
    cooRowIndexHostPtr[94]=23; cooColIndexHostPtr[94]=123; cooValHostPtr[94]=4.0;
    cooRowIndexHostPtr[95]=24; cooColIndexHostPtr[95]=23; cooValHostPtr[95]=310.0731361794753;
    cooRowIndexHostPtr[96]=24; cooColIndexHostPtr[96]=24; cooValHostPtr[96]=-615.6962723589505;
    cooRowIndexHostPtr[97]=24; cooColIndexHostPtr[97]=25; cooValHostPtr[97]=310.0731361794753;
    cooRowIndexHostPtr[98]=24; cooColIndexHostPtr[98]=124; cooValHostPtr[98]=4.0;
    cooRowIndexHostPtr[99]=25; cooColIndexHostPtr[99]=24; cooValHostPtr[99]=310.0731361794753;
    cooRowIndexHostPtr[100]=25; cooColIndexHostPtr[100]=25; cooValHostPtr[100]=-615.6962723589505;
    cooRowIndexHostPtr[101]=25; cooColIndexHostPtr[101]=26; cooValHostPtr[101]=310.0731361794753;
    cooRowIndexHostPtr[102]=25; cooColIndexHostPtr[102]=125; cooValHostPtr[102]=4.0;
    cooRowIndexHostPtr[103]=26; cooColIndexHostPtr[103]=25; cooValHostPtr[103]=310.0731361794753;
    cooRowIndexHostPtr[104]=26; cooColIndexHostPtr[104]=26; cooValHostPtr[104]=-615.6962723589505;
    cooRowIndexHostPtr[105]=26; cooColIndexHostPtr[105]=27; cooValHostPtr[105]=310.0731361794753;
    cooRowIndexHostPtr[106]=26; cooColIndexHostPtr[106]=126; cooValHostPtr[106]=4.0;
    cooRowIndexHostPtr[107]=27; cooColIndexHostPtr[107]=26; cooValHostPtr[107]=310.0731361794753;
    cooRowIndexHostPtr[108]=27; cooColIndexHostPtr[108]=27; cooValHostPtr[108]=-615.6962723589505;
    cooRowIndexHostPtr[109]=27; cooColIndexHostPtr[109]=28; cooValHostPtr[109]=310.0731361794753;
    cooRowIndexHostPtr[110]=27; cooColIndexHostPtr[110]=127; cooValHostPtr[110]=4.0;
    cooRowIndexHostPtr[111]=28; cooColIndexHostPtr[111]=27; cooValHostPtr[111]=310.0731361794753;
    cooRowIndexHostPtr[112]=28; cooColIndexHostPtr[112]=28; cooValHostPtr[112]=-615.6962723589505;
    cooRowIndexHostPtr[113]=28; cooColIndexHostPtr[113]=29; cooValHostPtr[113]=310.0731361794753;
    cooRowIndexHostPtr[114]=28; cooColIndexHostPtr[114]=128; cooValHostPtr[114]=4.0;
    cooRowIndexHostPtr[115]=29; cooColIndexHostPtr[115]=28; cooValHostPtr[115]=310.0731361794753;
    cooRowIndexHostPtr[116]=29; cooColIndexHostPtr[116]=29; cooValHostPtr[116]=-615.6962723589505;
    cooRowIndexHostPtr[117]=29; cooColIndexHostPtr[117]=30; cooValHostPtr[117]=310.0731361794753;
    cooRowIndexHostPtr[118]=29; cooColIndexHostPtr[118]=129; cooValHostPtr[118]=4.0;
    cooRowIndexHostPtr[119]=30; cooColIndexHostPtr[119]=29; cooValHostPtr[119]=310.0731361794753;
    cooRowIndexHostPtr[120]=30; cooColIndexHostPtr[120]=30; cooValHostPtr[120]=-615.6962723589505;
    cooRowIndexHostPtr[121]=30; cooColIndexHostPtr[121]=31; cooValHostPtr[121]=310.0731361794753;
    cooRowIndexHostPtr[122]=30; cooColIndexHostPtr[122]=130; cooValHostPtr[122]=4.0;
    cooRowIndexHostPtr[123]=31; cooColIndexHostPtr[123]=30; cooValHostPtr[123]=310.0731361794753;
    cooRowIndexHostPtr[124]=31; cooColIndexHostPtr[124]=31; cooValHostPtr[124]=-615.6962723589505;
    cooRowIndexHostPtr[125]=31; cooColIndexHostPtr[125]=32; cooValHostPtr[125]=310.0731361794753;
    cooRowIndexHostPtr[126]=31; cooColIndexHostPtr[126]=131; cooValHostPtr[126]=4.0;
    cooRowIndexHostPtr[127]=32; cooColIndexHostPtr[127]=31; cooValHostPtr[127]=310.0731361794753;
    cooRowIndexHostPtr[128]=32; cooColIndexHostPtr[128]=32; cooValHostPtr[128]=-615.6962723589505;
    cooRowIndexHostPtr[129]=32; cooColIndexHostPtr[129]=33; cooValHostPtr[129]=310.0731361794753;
    cooRowIndexHostPtr[130]=32; cooColIndexHostPtr[130]=132; cooValHostPtr[130]=4.0;
    cooRowIndexHostPtr[131]=33; cooColIndexHostPtr[131]=32; cooValHostPtr[131]=310.0731361794753;
    cooRowIndexHostPtr[132]=33; cooColIndexHostPtr[132]=33; cooValHostPtr[132]=-615.6962723589505;
    cooRowIndexHostPtr[133]=33; cooColIndexHostPtr[133]=34; cooValHostPtr[133]=310.0731361794753;
    cooRowIndexHostPtr[134]=33; cooColIndexHostPtr[134]=133; cooValHostPtr[134]=4.0;
    cooRowIndexHostPtr[135]=34; cooColIndexHostPtr[135]=33; cooValHostPtr[135]=310.0731361794753;
    cooRowIndexHostPtr[136]=34; cooColIndexHostPtr[136]=34; cooValHostPtr[136]=-615.6962723589505;
    cooRowIndexHostPtr[137]=34; cooColIndexHostPtr[137]=35; cooValHostPtr[137]=310.0731361794753;
    cooRowIndexHostPtr[138]=34; cooColIndexHostPtr[138]=134; cooValHostPtr[138]=4.0;
    cooRowIndexHostPtr[139]=35; cooColIndexHostPtr[139]=34; cooValHostPtr[139]=310.0731361794753;
    cooRowIndexHostPtr[140]=35; cooColIndexHostPtr[140]=35; cooValHostPtr[140]=-615.6962723589505;
    cooRowIndexHostPtr[141]=35; cooColIndexHostPtr[141]=36; cooValHostPtr[141]=310.0731361794753;
    cooRowIndexHostPtr[142]=35; cooColIndexHostPtr[142]=135; cooValHostPtr[142]=4.0;
    cooRowIndexHostPtr[143]=36; cooColIndexHostPtr[143]=35; cooValHostPtr[143]=310.0731361794753;
    cooRowIndexHostPtr[144]=36; cooColIndexHostPtr[144]=36; cooValHostPtr[144]=-615.6962723589505;
    cooRowIndexHostPtr[145]=36; cooColIndexHostPtr[145]=37; cooValHostPtr[145]=310.0731361794753;
    cooRowIndexHostPtr[146]=36; cooColIndexHostPtr[146]=136; cooValHostPtr[146]=4.0;
    cooRowIndexHostPtr[147]=37; cooColIndexHostPtr[147]=36; cooValHostPtr[147]=310.0731361794753;
    cooRowIndexHostPtr[148]=37; cooColIndexHostPtr[148]=37; cooValHostPtr[148]=-615.6962723589505;
    cooRowIndexHostPtr[149]=37; cooColIndexHostPtr[149]=38; cooValHostPtr[149]=310.0731361794753;
    cooRowIndexHostPtr[150]=37; cooColIndexHostPtr[150]=137; cooValHostPtr[150]=4.0;
    cooRowIndexHostPtr[151]=38; cooColIndexHostPtr[151]=37; cooValHostPtr[151]=310.0731361794753;
    cooRowIndexHostPtr[152]=38; cooColIndexHostPtr[152]=38; cooValHostPtr[152]=-615.6962723589505;
    cooRowIndexHostPtr[153]=38; cooColIndexHostPtr[153]=39; cooValHostPtr[153]=310.0731361794753;
    cooRowIndexHostPtr[154]=38; cooColIndexHostPtr[154]=138; cooValHostPtr[154]=4.0;
    cooRowIndexHostPtr[155]=39; cooColIndexHostPtr[155]=38; cooValHostPtr[155]=310.0731361794753;
    cooRowIndexHostPtr[156]=39; cooColIndexHostPtr[156]=39; cooValHostPtr[156]=-615.6962723589505;
    cooRowIndexHostPtr[157]=39; cooColIndexHostPtr[157]=40; cooValHostPtr[157]=310.0731361794753;
    cooRowIndexHostPtr[158]=39; cooColIndexHostPtr[158]=139; cooValHostPtr[158]=4.0;
    cooRowIndexHostPtr[159]=40; cooColIndexHostPtr[159]=39; cooValHostPtr[159]=310.0731361794753;
    cooRowIndexHostPtr[160]=40; cooColIndexHostPtr[160]=40; cooValHostPtr[160]=-615.6962723589505;
    cooRowIndexHostPtr[161]=40; cooColIndexHostPtr[161]=41; cooValHostPtr[161]=310.0731361794753;
    cooRowIndexHostPtr[162]=40; cooColIndexHostPtr[162]=140; cooValHostPtr[162]=4.0;
    cooRowIndexHostPtr[163]=41; cooColIndexHostPtr[163]=40; cooValHostPtr[163]=310.0731361794753;
    cooRowIndexHostPtr[164]=41; cooColIndexHostPtr[164]=41; cooValHostPtr[164]=-615.6962723589505;
    cooRowIndexHostPtr[165]=41; cooColIndexHostPtr[165]=42; cooValHostPtr[165]=310.0731361794753;
    cooRowIndexHostPtr[166]=41; cooColIndexHostPtr[166]=141; cooValHostPtr[166]=4.0;
    cooRowIndexHostPtr[167]=42; cooColIndexHostPtr[167]=41; cooValHostPtr[167]=310.0731361794753;
    cooRowIndexHostPtr[168]=42; cooColIndexHostPtr[168]=42; cooValHostPtr[168]=-615.6962723589505;
    cooRowIndexHostPtr[169]=42; cooColIndexHostPtr[169]=43; cooValHostPtr[169]=310.0731361794753;
    cooRowIndexHostPtr[170]=42; cooColIndexHostPtr[170]=142; cooValHostPtr[170]=4.0;
    cooRowIndexHostPtr[171]=43; cooColIndexHostPtr[171]=42; cooValHostPtr[171]=310.0731361794753;
    cooRowIndexHostPtr[172]=43; cooColIndexHostPtr[172]=43; cooValHostPtr[172]=-615.6962723589505;
    cooRowIndexHostPtr[173]=43; cooColIndexHostPtr[173]=44; cooValHostPtr[173]=310.0731361794753;
    cooRowIndexHostPtr[174]=43; cooColIndexHostPtr[174]=143; cooValHostPtr[174]=4.0;
    cooRowIndexHostPtr[175]=44; cooColIndexHostPtr[175]=43; cooValHostPtr[175]=310.0731361794753;
    cooRowIndexHostPtr[176]=44; cooColIndexHostPtr[176]=44; cooValHostPtr[176]=-615.6962723589505;
    cooRowIndexHostPtr[177]=44; cooColIndexHostPtr[177]=45; cooValHostPtr[177]=310.0731361794753;
    cooRowIndexHostPtr[178]=44; cooColIndexHostPtr[178]=144; cooValHostPtr[178]=4.0;
    cooRowIndexHostPtr[179]=45; cooColIndexHostPtr[179]=44; cooValHostPtr[179]=310.0731361794753;
    cooRowIndexHostPtr[180]=45; cooColIndexHostPtr[180]=45; cooValHostPtr[180]=-615.6962723589505;
    cooRowIndexHostPtr[181]=45; cooColIndexHostPtr[181]=46; cooValHostPtr[181]=310.0731361794753;
    cooRowIndexHostPtr[182]=45; cooColIndexHostPtr[182]=145; cooValHostPtr[182]=4.0;
    cooRowIndexHostPtr[183]=46; cooColIndexHostPtr[183]=45; cooValHostPtr[183]=310.0731361794753;
    cooRowIndexHostPtr[184]=46; cooColIndexHostPtr[184]=46; cooValHostPtr[184]=-615.6962723589505;
    cooRowIndexHostPtr[185]=46; cooColIndexHostPtr[185]=47; cooValHostPtr[185]=310.0731361794753;
    cooRowIndexHostPtr[186]=46; cooColIndexHostPtr[186]=146; cooValHostPtr[186]=4.0;
    cooRowIndexHostPtr[187]=47; cooColIndexHostPtr[187]=46; cooValHostPtr[187]=310.0731361794753;
    cooRowIndexHostPtr[188]=47; cooColIndexHostPtr[188]=47; cooValHostPtr[188]=-615.6962723589505;
    cooRowIndexHostPtr[189]=47; cooColIndexHostPtr[189]=48; cooValHostPtr[189]=310.0731361794753;
    cooRowIndexHostPtr[190]=47; cooColIndexHostPtr[190]=147; cooValHostPtr[190]=4.0;
    cooRowIndexHostPtr[191]=48; cooColIndexHostPtr[191]=47; cooValHostPtr[191]=310.0731361794753;
    cooRowIndexHostPtr[192]=48; cooColIndexHostPtr[192]=48; cooValHostPtr[192]=-615.6962723589505;
    cooRowIndexHostPtr[193]=48; cooColIndexHostPtr[193]=49; cooValHostPtr[193]=310.0731361794753;
    cooRowIndexHostPtr[194]=48; cooColIndexHostPtr[194]=148; cooValHostPtr[194]=4.0;
    cooRowIndexHostPtr[195]=49; cooColIndexHostPtr[195]=48; cooValHostPtr[195]=310.0731361794753;
    cooRowIndexHostPtr[196]=49; cooColIndexHostPtr[196]=49; cooValHostPtr[196]=-615.6962723589505;
    cooRowIndexHostPtr[197]=49; cooColIndexHostPtr[197]=50; cooValHostPtr[197]=310.0731361794753;
    cooRowIndexHostPtr[198]=49; cooColIndexHostPtr[198]=149; cooValHostPtr[198]=4.0;
    cooRowIndexHostPtr[199]=50; cooColIndexHostPtr[199]=49; cooValHostPtr[199]=310.0731361794753;
    cooRowIndexHostPtr[200]=50; cooColIndexHostPtr[200]=50; cooValHostPtr[200]=-615.6962723589505;
    cooRowIndexHostPtr[201]=50; cooColIndexHostPtr[201]=51; cooValHostPtr[201]=310.0731361794753;
    cooRowIndexHostPtr[202]=50; cooColIndexHostPtr[202]=150; cooValHostPtr[202]=4.0;
    cooRowIndexHostPtr[203]=51; cooColIndexHostPtr[203]=50; cooValHostPtr[203]=310.0731361794753;
    cooRowIndexHostPtr[204]=51; cooColIndexHostPtr[204]=51; cooValHostPtr[204]=-615.6962723589505;
    cooRowIndexHostPtr[205]=51; cooColIndexHostPtr[205]=52; cooValHostPtr[205]=310.0731361794753;
    cooRowIndexHostPtr[206]=51; cooColIndexHostPtr[206]=151; cooValHostPtr[206]=4.0;
    cooRowIndexHostPtr[207]=52; cooColIndexHostPtr[207]=51; cooValHostPtr[207]=310.0731361794753;
    cooRowIndexHostPtr[208]=52; cooColIndexHostPtr[208]=52; cooValHostPtr[208]=-615.6962723589505;
    cooRowIndexHostPtr[209]=52; cooColIndexHostPtr[209]=53; cooValHostPtr[209]=310.0731361794753;
    cooRowIndexHostPtr[210]=52; cooColIndexHostPtr[210]=152; cooValHostPtr[210]=4.0;
    cooRowIndexHostPtr[211]=53; cooColIndexHostPtr[211]=52; cooValHostPtr[211]=310.0731361794753;
    cooRowIndexHostPtr[212]=53; cooColIndexHostPtr[212]=53; cooValHostPtr[212]=-615.6962723589505;
    cooRowIndexHostPtr[213]=53; cooColIndexHostPtr[213]=54; cooValHostPtr[213]=310.0731361794753;
    cooRowIndexHostPtr[214]=53; cooColIndexHostPtr[214]=153; cooValHostPtr[214]=4.0;
    cooRowIndexHostPtr[215]=54; cooColIndexHostPtr[215]=53; cooValHostPtr[215]=310.0731361794753;
    cooRowIndexHostPtr[216]=54; cooColIndexHostPtr[216]=54; cooValHostPtr[216]=-615.6962723589505;
    cooRowIndexHostPtr[217]=54; cooColIndexHostPtr[217]=55; cooValHostPtr[217]=310.0731361794753;
    cooRowIndexHostPtr[218]=54; cooColIndexHostPtr[218]=154; cooValHostPtr[218]=4.0;
    cooRowIndexHostPtr[219]=55; cooColIndexHostPtr[219]=54; cooValHostPtr[219]=310.0731361794753;
    cooRowIndexHostPtr[220]=55; cooColIndexHostPtr[220]=55; cooValHostPtr[220]=-615.6962723589505;
    cooRowIndexHostPtr[221]=55; cooColIndexHostPtr[221]=56; cooValHostPtr[221]=310.0731361794753;
    cooRowIndexHostPtr[222]=55; cooColIndexHostPtr[222]=155; cooValHostPtr[222]=4.0;
    cooRowIndexHostPtr[223]=56; cooColIndexHostPtr[223]=55; cooValHostPtr[223]=310.0731361794753;
    cooRowIndexHostPtr[224]=56; cooColIndexHostPtr[224]=56; cooValHostPtr[224]=-615.6962723589505;
    cooRowIndexHostPtr[225]=56; cooColIndexHostPtr[225]=57; cooValHostPtr[225]=310.0731361794753;
    cooRowIndexHostPtr[226]=56; cooColIndexHostPtr[226]=156; cooValHostPtr[226]=4.0;
    cooRowIndexHostPtr[227]=57; cooColIndexHostPtr[227]=56; cooValHostPtr[227]=310.0731361794753;
    cooRowIndexHostPtr[228]=57; cooColIndexHostPtr[228]=57; cooValHostPtr[228]=-615.6962723589505;
    cooRowIndexHostPtr[229]=57; cooColIndexHostPtr[229]=58; cooValHostPtr[229]=310.0731361794753;
    cooRowIndexHostPtr[230]=57; cooColIndexHostPtr[230]=157; cooValHostPtr[230]=4.0;
    cooRowIndexHostPtr[231]=58; cooColIndexHostPtr[231]=57; cooValHostPtr[231]=310.0731361794753;
    cooRowIndexHostPtr[232]=58; cooColIndexHostPtr[232]=58; cooValHostPtr[232]=-615.6962723589505;
    cooRowIndexHostPtr[233]=58; cooColIndexHostPtr[233]=59; cooValHostPtr[233]=310.0731361794753;
    cooRowIndexHostPtr[234]=58; cooColIndexHostPtr[234]=158; cooValHostPtr[234]=4.0;
    cooRowIndexHostPtr[235]=59; cooColIndexHostPtr[235]=58; cooValHostPtr[235]=310.0731361794753;
    cooRowIndexHostPtr[236]=59; cooColIndexHostPtr[236]=59; cooValHostPtr[236]=-615.6962723589505;
    cooRowIndexHostPtr[237]=59; cooColIndexHostPtr[237]=60; cooValHostPtr[237]=310.0731361794753;
    cooRowIndexHostPtr[238]=59; cooColIndexHostPtr[238]=159; cooValHostPtr[238]=4.0;
    cooRowIndexHostPtr[239]=60; cooColIndexHostPtr[239]=59; cooValHostPtr[239]=310.0731361794753;
    cooRowIndexHostPtr[240]=60; cooColIndexHostPtr[240]=60; cooValHostPtr[240]=-615.6962723589505;
    cooRowIndexHostPtr[241]=60; cooColIndexHostPtr[241]=61; cooValHostPtr[241]=310.0731361794753;
    cooRowIndexHostPtr[242]=60; cooColIndexHostPtr[242]=160; cooValHostPtr[242]=4.0;
    cooRowIndexHostPtr[243]=61; cooColIndexHostPtr[243]=60; cooValHostPtr[243]=310.0731361794753;
    cooRowIndexHostPtr[244]=61; cooColIndexHostPtr[244]=61; cooValHostPtr[244]=-615.6962723589505;
    cooRowIndexHostPtr[245]=61; cooColIndexHostPtr[245]=62; cooValHostPtr[245]=310.0731361794753;
    cooRowIndexHostPtr[246]=61; cooColIndexHostPtr[246]=161; cooValHostPtr[246]=4.0;
    cooRowIndexHostPtr[247]=62; cooColIndexHostPtr[247]=61; cooValHostPtr[247]=310.0731361794753;
    cooRowIndexHostPtr[248]=62; cooColIndexHostPtr[248]=62; cooValHostPtr[248]=-615.6962723589505;
    cooRowIndexHostPtr[249]=62; cooColIndexHostPtr[249]=63; cooValHostPtr[249]=310.0731361794753;
    cooRowIndexHostPtr[250]=62; cooColIndexHostPtr[250]=162; cooValHostPtr[250]=4.0;
    cooRowIndexHostPtr[251]=63; cooColIndexHostPtr[251]=62; cooValHostPtr[251]=310.0731361794753;
    cooRowIndexHostPtr[252]=63; cooColIndexHostPtr[252]=63; cooValHostPtr[252]=-615.6962723589505;
    cooRowIndexHostPtr[253]=63; cooColIndexHostPtr[253]=64; cooValHostPtr[253]=310.0731361794753;
    cooRowIndexHostPtr[254]=63; cooColIndexHostPtr[254]=163; cooValHostPtr[254]=4.0;
    cooRowIndexHostPtr[255]=64; cooColIndexHostPtr[255]=63; cooValHostPtr[255]=310.0731361794753;
    cooRowIndexHostPtr[256]=64; cooColIndexHostPtr[256]=64; cooValHostPtr[256]=-615.6962723589505;
    cooRowIndexHostPtr[257]=64; cooColIndexHostPtr[257]=65; cooValHostPtr[257]=310.0731361794753;
    cooRowIndexHostPtr[258]=64; cooColIndexHostPtr[258]=164; cooValHostPtr[258]=4.0;
    cooRowIndexHostPtr[259]=65; cooColIndexHostPtr[259]=64; cooValHostPtr[259]=310.0731361794753;
    cooRowIndexHostPtr[260]=65; cooColIndexHostPtr[260]=65; cooValHostPtr[260]=-615.6962723589505;
    cooRowIndexHostPtr[261]=65; cooColIndexHostPtr[261]=66; cooValHostPtr[261]=310.0731361794753;
    cooRowIndexHostPtr[262]=65; cooColIndexHostPtr[262]=165; cooValHostPtr[262]=4.0;
    cooRowIndexHostPtr[263]=66; cooColIndexHostPtr[263]=65; cooValHostPtr[263]=310.0731361794753;
    cooRowIndexHostPtr[264]=66; cooColIndexHostPtr[264]=66; cooValHostPtr[264]=-615.6962723589505;
    cooRowIndexHostPtr[265]=66; cooColIndexHostPtr[265]=67; cooValHostPtr[265]=310.0731361794753;
    cooRowIndexHostPtr[266]=66; cooColIndexHostPtr[266]=166; cooValHostPtr[266]=4.0;
    cooRowIndexHostPtr[267]=67; cooColIndexHostPtr[267]=66; cooValHostPtr[267]=310.0731361794753;
    cooRowIndexHostPtr[268]=67; cooColIndexHostPtr[268]=67; cooValHostPtr[268]=-615.6962723589505;
    cooRowIndexHostPtr[269]=67; cooColIndexHostPtr[269]=68; cooValHostPtr[269]=310.0731361794753;
    cooRowIndexHostPtr[270]=67; cooColIndexHostPtr[270]=167; cooValHostPtr[270]=4.0;
    cooRowIndexHostPtr[271]=68; cooColIndexHostPtr[271]=67; cooValHostPtr[271]=310.0731361794753;
    cooRowIndexHostPtr[272]=68; cooColIndexHostPtr[272]=68; cooValHostPtr[272]=-615.6962723589505;
    cooRowIndexHostPtr[273]=68; cooColIndexHostPtr[273]=69; cooValHostPtr[273]=310.0731361794753;
    cooRowIndexHostPtr[274]=68; cooColIndexHostPtr[274]=168; cooValHostPtr[274]=4.0;
    cooRowIndexHostPtr[275]=69; cooColIndexHostPtr[275]=68; cooValHostPtr[275]=310.0731361794753;
    cooRowIndexHostPtr[276]=69; cooColIndexHostPtr[276]=69; cooValHostPtr[276]=-615.6962723589505;
    cooRowIndexHostPtr[277]=69; cooColIndexHostPtr[277]=70; cooValHostPtr[277]=310.0731361794753;
    cooRowIndexHostPtr[278]=69; cooColIndexHostPtr[278]=169; cooValHostPtr[278]=4.0;
    cooRowIndexHostPtr[279]=70; cooColIndexHostPtr[279]=69; cooValHostPtr[279]=310.0731361794753;
    cooRowIndexHostPtr[280]=70; cooColIndexHostPtr[280]=70; cooValHostPtr[280]=-615.6962723589505;
    cooRowIndexHostPtr[281]=70; cooColIndexHostPtr[281]=71; cooValHostPtr[281]=310.0731361794753;
    cooRowIndexHostPtr[282]=70; cooColIndexHostPtr[282]=170; cooValHostPtr[282]=4.0;
    cooRowIndexHostPtr[283]=71; cooColIndexHostPtr[283]=70; cooValHostPtr[283]=310.0731361794753;
    cooRowIndexHostPtr[284]=71; cooColIndexHostPtr[284]=71; cooValHostPtr[284]=-615.6962723589505;
    cooRowIndexHostPtr[285]=71; cooColIndexHostPtr[285]=72; cooValHostPtr[285]=310.0731361794753;
    cooRowIndexHostPtr[286]=71; cooColIndexHostPtr[286]=171; cooValHostPtr[286]=4.0;
    cooRowIndexHostPtr[287]=72; cooColIndexHostPtr[287]=71; cooValHostPtr[287]=310.0731361794753;
    cooRowIndexHostPtr[288]=72; cooColIndexHostPtr[288]=72; cooValHostPtr[288]=-615.6962723589505;
    cooRowIndexHostPtr[289]=72; cooColIndexHostPtr[289]=73; cooValHostPtr[289]=310.0731361794753;
    cooRowIndexHostPtr[290]=72; cooColIndexHostPtr[290]=172; cooValHostPtr[290]=4.0;
    cooRowIndexHostPtr[291]=73; cooColIndexHostPtr[291]=72; cooValHostPtr[291]=310.0731361794753;
    cooRowIndexHostPtr[292]=73; cooColIndexHostPtr[292]=73; cooValHostPtr[292]=-615.6962723589505;
    cooRowIndexHostPtr[293]=73; cooColIndexHostPtr[293]=74; cooValHostPtr[293]=310.0731361794753;
    cooRowIndexHostPtr[294]=73; cooColIndexHostPtr[294]=173; cooValHostPtr[294]=4.0;
    cooRowIndexHostPtr[295]=74; cooColIndexHostPtr[295]=73; cooValHostPtr[295]=310.0731361794753;
    cooRowIndexHostPtr[296]=74; cooColIndexHostPtr[296]=74; cooValHostPtr[296]=-615.6962723589505;
    cooRowIndexHostPtr[297]=74; cooColIndexHostPtr[297]=75; cooValHostPtr[297]=310.0731361794753;
    cooRowIndexHostPtr[298]=74; cooColIndexHostPtr[298]=174; cooValHostPtr[298]=4.0;
    cooRowIndexHostPtr[299]=75; cooColIndexHostPtr[299]=74; cooValHostPtr[299]=310.0731361794753;
    cooRowIndexHostPtr[300]=75; cooColIndexHostPtr[300]=75; cooValHostPtr[300]=-615.6962723589505;
    cooRowIndexHostPtr[301]=75; cooColIndexHostPtr[301]=76; cooValHostPtr[301]=310.0731361794753;
    cooRowIndexHostPtr[302]=75; cooColIndexHostPtr[302]=175; cooValHostPtr[302]=4.0;
    cooRowIndexHostPtr[303]=76; cooColIndexHostPtr[303]=75; cooValHostPtr[303]=310.0731361794753;
    cooRowIndexHostPtr[304]=76; cooColIndexHostPtr[304]=76; cooValHostPtr[304]=-615.6962723589505;
    cooRowIndexHostPtr[305]=76; cooColIndexHostPtr[305]=77; cooValHostPtr[305]=310.0731361794753;
    cooRowIndexHostPtr[306]=76; cooColIndexHostPtr[306]=176; cooValHostPtr[306]=4.0;
    cooRowIndexHostPtr[307]=77; cooColIndexHostPtr[307]=76; cooValHostPtr[307]=310.0731361794753;
    cooRowIndexHostPtr[308]=77; cooColIndexHostPtr[308]=77; cooValHostPtr[308]=-615.6962723589505;
    cooRowIndexHostPtr[309]=77; cooColIndexHostPtr[309]=78; cooValHostPtr[309]=310.0731361794753;
    cooRowIndexHostPtr[310]=77; cooColIndexHostPtr[310]=177; cooValHostPtr[310]=4.0;
    cooRowIndexHostPtr[311]=78; cooColIndexHostPtr[311]=77; cooValHostPtr[311]=310.0731361794753;
    cooRowIndexHostPtr[312]=78; cooColIndexHostPtr[312]=78; cooValHostPtr[312]=-615.6962723589505;
    cooRowIndexHostPtr[313]=78; cooColIndexHostPtr[313]=79; cooValHostPtr[313]=310.0731361794753;
    cooRowIndexHostPtr[314]=78; cooColIndexHostPtr[314]=178; cooValHostPtr[314]=4.0;
    cooRowIndexHostPtr[315]=79; cooColIndexHostPtr[315]=78; cooValHostPtr[315]=310.0731361794753;
    cooRowIndexHostPtr[316]=79; cooColIndexHostPtr[316]=79; cooValHostPtr[316]=-615.6962723589505;
    cooRowIndexHostPtr[317]=79; cooColIndexHostPtr[317]=80; cooValHostPtr[317]=310.0731361794753;
    cooRowIndexHostPtr[318]=79; cooColIndexHostPtr[318]=179; cooValHostPtr[318]=4.0;
    cooRowIndexHostPtr[319]=80; cooColIndexHostPtr[319]=79; cooValHostPtr[319]=310.0731361794753;
    cooRowIndexHostPtr[320]=80; cooColIndexHostPtr[320]=80; cooValHostPtr[320]=-615.6962723589505;
    cooRowIndexHostPtr[321]=80; cooColIndexHostPtr[321]=81; cooValHostPtr[321]=310.0731361794753;
    cooRowIndexHostPtr[322]=80; cooColIndexHostPtr[322]=180; cooValHostPtr[322]=4.0;
    cooRowIndexHostPtr[323]=81; cooColIndexHostPtr[323]=80; cooValHostPtr[323]=310.0731361794753;
    cooRowIndexHostPtr[324]=81; cooColIndexHostPtr[324]=81; cooValHostPtr[324]=-615.6962723589505;
    cooRowIndexHostPtr[325]=81; cooColIndexHostPtr[325]=82; cooValHostPtr[325]=310.0731361794753;
    cooRowIndexHostPtr[326]=81; cooColIndexHostPtr[326]=181; cooValHostPtr[326]=4.0;
    cooRowIndexHostPtr[327]=82; cooColIndexHostPtr[327]=81; cooValHostPtr[327]=310.0731361794753;
    cooRowIndexHostPtr[328]=82; cooColIndexHostPtr[328]=82; cooValHostPtr[328]=-615.6962723589505;
    cooRowIndexHostPtr[329]=82; cooColIndexHostPtr[329]=83; cooValHostPtr[329]=310.0731361794753;
    cooRowIndexHostPtr[330]=82; cooColIndexHostPtr[330]=182; cooValHostPtr[330]=4.0;
    cooRowIndexHostPtr[331]=83; cooColIndexHostPtr[331]=82; cooValHostPtr[331]=310.0731361794753;
    cooRowIndexHostPtr[332]=83; cooColIndexHostPtr[332]=83; cooValHostPtr[332]=-615.6962723589505;
    cooRowIndexHostPtr[333]=83; cooColIndexHostPtr[333]=84; cooValHostPtr[333]=310.0731361794753;
    cooRowIndexHostPtr[334]=83; cooColIndexHostPtr[334]=183; cooValHostPtr[334]=4.0;
    cooRowIndexHostPtr[335]=84; cooColIndexHostPtr[335]=83; cooValHostPtr[335]=310.0731361794753;
    cooRowIndexHostPtr[336]=84; cooColIndexHostPtr[336]=84; cooValHostPtr[336]=-615.6962723589505;
    cooRowIndexHostPtr[337]=84; cooColIndexHostPtr[337]=85; cooValHostPtr[337]=310.0731361794753;
    cooRowIndexHostPtr[338]=84; cooColIndexHostPtr[338]=184; cooValHostPtr[338]=4.0;
    cooRowIndexHostPtr[339]=85; cooColIndexHostPtr[339]=84; cooValHostPtr[339]=310.0731361794753;
    cooRowIndexHostPtr[340]=85; cooColIndexHostPtr[340]=85; cooValHostPtr[340]=-615.6962723589505;
    cooRowIndexHostPtr[341]=85; cooColIndexHostPtr[341]=86; cooValHostPtr[341]=310.0731361794753;
    cooRowIndexHostPtr[342]=85; cooColIndexHostPtr[342]=185; cooValHostPtr[342]=4.0;
    cooRowIndexHostPtr[343]=86; cooColIndexHostPtr[343]=85; cooValHostPtr[343]=310.0731361794753;
    cooRowIndexHostPtr[344]=86; cooColIndexHostPtr[344]=86; cooValHostPtr[344]=-615.6962723589505;
    cooRowIndexHostPtr[345]=86; cooColIndexHostPtr[345]=87; cooValHostPtr[345]=310.0731361794753;
    cooRowIndexHostPtr[346]=86; cooColIndexHostPtr[346]=186; cooValHostPtr[346]=4.0;
    cooRowIndexHostPtr[347]=87; cooColIndexHostPtr[347]=86; cooValHostPtr[347]=310.0731361794753;
    cooRowIndexHostPtr[348]=87; cooColIndexHostPtr[348]=87; cooValHostPtr[348]=-615.6962723589505;
    cooRowIndexHostPtr[349]=87; cooColIndexHostPtr[349]=88; cooValHostPtr[349]=310.0731361794753;
    cooRowIndexHostPtr[350]=87; cooColIndexHostPtr[350]=187; cooValHostPtr[350]=4.0;
    cooRowIndexHostPtr[351]=88; cooColIndexHostPtr[351]=87; cooValHostPtr[351]=310.0731361794753;
    cooRowIndexHostPtr[352]=88; cooColIndexHostPtr[352]=88; cooValHostPtr[352]=-615.6962723589505;
    cooRowIndexHostPtr[353]=88; cooColIndexHostPtr[353]=89; cooValHostPtr[353]=310.0731361794753;
    cooRowIndexHostPtr[354]=88; cooColIndexHostPtr[354]=188; cooValHostPtr[354]=4.0;
    cooRowIndexHostPtr[355]=89; cooColIndexHostPtr[355]=88; cooValHostPtr[355]=310.0731361794753;
    cooRowIndexHostPtr[356]=89; cooColIndexHostPtr[356]=89; cooValHostPtr[356]=-615.6962723589505;
    cooRowIndexHostPtr[357]=89; cooColIndexHostPtr[357]=90; cooValHostPtr[357]=310.0731361794753;
    cooRowIndexHostPtr[358]=89; cooColIndexHostPtr[358]=189; cooValHostPtr[358]=4.0;
    cooRowIndexHostPtr[359]=90; cooColIndexHostPtr[359]=89; cooValHostPtr[359]=310.0731361794753;
    cooRowIndexHostPtr[360]=90; cooColIndexHostPtr[360]=90; cooValHostPtr[360]=-615.6962723589505;
    cooRowIndexHostPtr[361]=90; cooColIndexHostPtr[361]=91; cooValHostPtr[361]=310.0731361794753;
    cooRowIndexHostPtr[362]=90; cooColIndexHostPtr[362]=190; cooValHostPtr[362]=4.0;
    cooRowIndexHostPtr[363]=91; cooColIndexHostPtr[363]=90; cooValHostPtr[363]=310.0731361794753;
    cooRowIndexHostPtr[364]=91; cooColIndexHostPtr[364]=91; cooValHostPtr[364]=-615.6962723589505;
    cooRowIndexHostPtr[365]=91; cooColIndexHostPtr[365]=92; cooValHostPtr[365]=310.0731361794753;
    cooRowIndexHostPtr[366]=91; cooColIndexHostPtr[366]=191; cooValHostPtr[366]=4.0;
    cooRowIndexHostPtr[367]=92; cooColIndexHostPtr[367]=91; cooValHostPtr[367]=310.0731361794753;
    cooRowIndexHostPtr[368]=92; cooColIndexHostPtr[368]=92; cooValHostPtr[368]=-615.6962723589505;
    cooRowIndexHostPtr[369]=92; cooColIndexHostPtr[369]=93; cooValHostPtr[369]=310.0731361794753;
    cooRowIndexHostPtr[370]=92; cooColIndexHostPtr[370]=192; cooValHostPtr[370]=4.0;
    cooRowIndexHostPtr[371]=93; cooColIndexHostPtr[371]=92; cooValHostPtr[371]=310.0731361794753;
    cooRowIndexHostPtr[372]=93; cooColIndexHostPtr[372]=93; cooValHostPtr[372]=-615.6962723589505;
    cooRowIndexHostPtr[373]=93; cooColIndexHostPtr[373]=94; cooValHostPtr[373]=310.0731361794753;
    cooRowIndexHostPtr[374]=93; cooColIndexHostPtr[374]=193; cooValHostPtr[374]=4.0;
    cooRowIndexHostPtr[375]=94; cooColIndexHostPtr[375]=93; cooValHostPtr[375]=310.0731361794753;
    cooRowIndexHostPtr[376]=94; cooColIndexHostPtr[376]=94; cooValHostPtr[376]=-615.6962723589505;
    cooRowIndexHostPtr[377]=94; cooColIndexHostPtr[377]=95; cooValHostPtr[377]=310.0731361794753;
    cooRowIndexHostPtr[378]=94; cooColIndexHostPtr[378]=194; cooValHostPtr[378]=4.0;
    cooRowIndexHostPtr[379]=95; cooColIndexHostPtr[379]=94; cooValHostPtr[379]=310.0731361794753;
    cooRowIndexHostPtr[380]=95; cooColIndexHostPtr[380]=95; cooValHostPtr[380]=-615.6962723589505;
    cooRowIndexHostPtr[381]=95; cooColIndexHostPtr[381]=96; cooValHostPtr[381]=310.0731361794753;
    cooRowIndexHostPtr[382]=95; cooColIndexHostPtr[382]=195; cooValHostPtr[382]=4.0;
    cooRowIndexHostPtr[383]=96; cooColIndexHostPtr[383]=95; cooValHostPtr[383]=310.0731361794753;
    cooRowIndexHostPtr[384]=96; cooColIndexHostPtr[384]=96; cooValHostPtr[384]=-615.6962723589505;
    cooRowIndexHostPtr[385]=96; cooColIndexHostPtr[385]=97; cooValHostPtr[385]=310.0731361794753;
    cooRowIndexHostPtr[386]=96; cooColIndexHostPtr[386]=196; cooValHostPtr[386]=4.0;
    cooRowIndexHostPtr[387]=97; cooColIndexHostPtr[387]=96; cooValHostPtr[387]=310.0731361794753;
    cooRowIndexHostPtr[388]=97; cooColIndexHostPtr[388]=97; cooValHostPtr[388]=-615.6962723589505;
    cooRowIndexHostPtr[389]=97; cooColIndexHostPtr[389]=98; cooValHostPtr[389]=310.0731361794753;
    cooRowIndexHostPtr[390]=97; cooColIndexHostPtr[390]=197; cooValHostPtr[390]=4.0;
    cooRowIndexHostPtr[391]=98; cooColIndexHostPtr[391]=97; cooValHostPtr[391]=310.0731361794753;
    cooRowIndexHostPtr[392]=98; cooColIndexHostPtr[392]=98; cooValHostPtr[392]=-615.6962723589505;
    cooRowIndexHostPtr[393]=98; cooColIndexHostPtr[393]=99; cooValHostPtr[393]=310.0731361794753;
    cooRowIndexHostPtr[394]=98; cooColIndexHostPtr[394]=198; cooValHostPtr[394]=4.0;
    cooRowIndexHostPtr[395]=99; cooColIndexHostPtr[395]=98; cooValHostPtr[395]=310.0731361794753;
    cooRowIndexHostPtr[396]=99; cooColIndexHostPtr[396]=99; cooValHostPtr[396]=-615.6962723589505;
    cooRowIndexHostPtr[397]=99; cooColIndexHostPtr[397]=199; cooValHostPtr[397]=4.0;
    cooRowIndexHostPtr[398]=100; cooColIndexHostPtr[398]=0; cooValHostPtr[398]=-5.45;
    cooRowIndexHostPtr[399]=100; cooColIndexHostPtr[399]=100; cooValHostPtr[399]=-314.0731361794753;
    cooRowIndexHostPtr[400]=100; cooColIndexHostPtr[400]=101; cooValHostPtr[400]=155.0365680897376;
    cooRowIndexHostPtr[401]=101; cooColIndexHostPtr[401]=1; cooValHostPtr[401]=-5.45;
    cooRowIndexHostPtr[402]=101; cooColIndexHostPtr[402]=100; cooValHostPtr[402]=155.0365680897376;
    cooRowIndexHostPtr[403]=101; cooColIndexHostPtr[403]=101; cooValHostPtr[403]=-314.0731361794753;
    cooRowIndexHostPtr[404]=101; cooColIndexHostPtr[404]=102; cooValHostPtr[404]=155.0365680897376;
    cooRowIndexHostPtr[405]=102; cooColIndexHostPtr[405]=2; cooValHostPtr[405]=-5.45;
    cooRowIndexHostPtr[406]=102; cooColIndexHostPtr[406]=101; cooValHostPtr[406]=155.0365680897376;
    cooRowIndexHostPtr[407]=102; cooColIndexHostPtr[407]=102; cooValHostPtr[407]=-314.0731361794753;
    cooRowIndexHostPtr[408]=102; cooColIndexHostPtr[408]=103; cooValHostPtr[408]=155.0365680897376;
    cooRowIndexHostPtr[409]=103; cooColIndexHostPtr[409]=3; cooValHostPtr[409]=-5.45;
    cooRowIndexHostPtr[410]=103; cooColIndexHostPtr[410]=102; cooValHostPtr[410]=155.0365680897376;
    cooRowIndexHostPtr[411]=103; cooColIndexHostPtr[411]=103; cooValHostPtr[411]=-314.0731361794753;
    cooRowIndexHostPtr[412]=103; cooColIndexHostPtr[412]=104; cooValHostPtr[412]=155.0365680897376;
    cooRowIndexHostPtr[413]=104; cooColIndexHostPtr[413]=4; cooValHostPtr[413]=-5.45;
    cooRowIndexHostPtr[414]=104; cooColIndexHostPtr[414]=103; cooValHostPtr[414]=155.0365680897376;
    cooRowIndexHostPtr[415]=104; cooColIndexHostPtr[415]=104; cooValHostPtr[415]=-314.0731361794753;
    cooRowIndexHostPtr[416]=104; cooColIndexHostPtr[416]=105; cooValHostPtr[416]=155.0365680897376;
    cooRowIndexHostPtr[417]=105; cooColIndexHostPtr[417]=5; cooValHostPtr[417]=-5.45;
    cooRowIndexHostPtr[418]=105; cooColIndexHostPtr[418]=104; cooValHostPtr[418]=155.0365680897376;
    cooRowIndexHostPtr[419]=105; cooColIndexHostPtr[419]=105; cooValHostPtr[419]=-314.0731361794753;
    cooRowIndexHostPtr[420]=105; cooColIndexHostPtr[420]=106; cooValHostPtr[420]=155.0365680897376;
    cooRowIndexHostPtr[421]=106; cooColIndexHostPtr[421]=6; cooValHostPtr[421]=-5.45;
    cooRowIndexHostPtr[422]=106; cooColIndexHostPtr[422]=105; cooValHostPtr[422]=155.0365680897376;
    cooRowIndexHostPtr[423]=106; cooColIndexHostPtr[423]=106; cooValHostPtr[423]=-314.0731361794753;
    cooRowIndexHostPtr[424]=106; cooColIndexHostPtr[424]=107; cooValHostPtr[424]=155.0365680897376;
    cooRowIndexHostPtr[425]=107; cooColIndexHostPtr[425]=7; cooValHostPtr[425]=-5.45;
    cooRowIndexHostPtr[426]=107; cooColIndexHostPtr[426]=106; cooValHostPtr[426]=155.0365680897376;
    cooRowIndexHostPtr[427]=107; cooColIndexHostPtr[427]=107; cooValHostPtr[427]=-314.0731361794753;
    cooRowIndexHostPtr[428]=107; cooColIndexHostPtr[428]=108; cooValHostPtr[428]=155.0365680897376;
    cooRowIndexHostPtr[429]=108; cooColIndexHostPtr[429]=8; cooValHostPtr[429]=-5.45;
    cooRowIndexHostPtr[430]=108; cooColIndexHostPtr[430]=107; cooValHostPtr[430]=155.0365680897376;
    cooRowIndexHostPtr[431]=108; cooColIndexHostPtr[431]=108; cooValHostPtr[431]=-314.0731361794753;
    cooRowIndexHostPtr[432]=108; cooColIndexHostPtr[432]=109; cooValHostPtr[432]=155.0365680897376;
    cooRowIndexHostPtr[433]=109; cooColIndexHostPtr[433]=9; cooValHostPtr[433]=-5.45;
    cooRowIndexHostPtr[434]=109; cooColIndexHostPtr[434]=108; cooValHostPtr[434]=155.0365680897376;
    cooRowIndexHostPtr[435]=109; cooColIndexHostPtr[435]=109; cooValHostPtr[435]=-314.0731361794753;
    cooRowIndexHostPtr[436]=109; cooColIndexHostPtr[436]=110; cooValHostPtr[436]=155.0365680897376;
    cooRowIndexHostPtr[437]=110; cooColIndexHostPtr[437]=10; cooValHostPtr[437]=-5.45;
    cooRowIndexHostPtr[438]=110; cooColIndexHostPtr[438]=109; cooValHostPtr[438]=155.0365680897376;
    cooRowIndexHostPtr[439]=110; cooColIndexHostPtr[439]=110; cooValHostPtr[439]=-314.0731361794753;
    cooRowIndexHostPtr[440]=110; cooColIndexHostPtr[440]=111; cooValHostPtr[440]=155.0365680897376;
    cooRowIndexHostPtr[441]=111; cooColIndexHostPtr[441]=11; cooValHostPtr[441]=-5.45;
    cooRowIndexHostPtr[442]=111; cooColIndexHostPtr[442]=110; cooValHostPtr[442]=155.0365680897376;
    cooRowIndexHostPtr[443]=111; cooColIndexHostPtr[443]=111; cooValHostPtr[443]=-314.0731361794753;
    cooRowIndexHostPtr[444]=111; cooColIndexHostPtr[444]=112; cooValHostPtr[444]=155.0365680897376;
    cooRowIndexHostPtr[445]=112; cooColIndexHostPtr[445]=12; cooValHostPtr[445]=-5.45;
    cooRowIndexHostPtr[446]=112; cooColIndexHostPtr[446]=111; cooValHostPtr[446]=155.0365680897376;
    cooRowIndexHostPtr[447]=112; cooColIndexHostPtr[447]=112; cooValHostPtr[447]=-314.0731361794753;
    cooRowIndexHostPtr[448]=112; cooColIndexHostPtr[448]=113; cooValHostPtr[448]=155.0365680897376;
    cooRowIndexHostPtr[449]=113; cooColIndexHostPtr[449]=13; cooValHostPtr[449]=-5.45;
    cooRowIndexHostPtr[450]=113; cooColIndexHostPtr[450]=112; cooValHostPtr[450]=155.0365680897376;
    cooRowIndexHostPtr[451]=113; cooColIndexHostPtr[451]=113; cooValHostPtr[451]=-314.0731361794753;
    cooRowIndexHostPtr[452]=113; cooColIndexHostPtr[452]=114; cooValHostPtr[452]=155.0365680897376;
    cooRowIndexHostPtr[453]=114; cooColIndexHostPtr[453]=14; cooValHostPtr[453]=-5.45;
    cooRowIndexHostPtr[454]=114; cooColIndexHostPtr[454]=113; cooValHostPtr[454]=155.0365680897376;
    cooRowIndexHostPtr[455]=114; cooColIndexHostPtr[455]=114; cooValHostPtr[455]=-314.0731361794753;
    cooRowIndexHostPtr[456]=114; cooColIndexHostPtr[456]=115; cooValHostPtr[456]=155.0365680897376;
    cooRowIndexHostPtr[457]=115; cooColIndexHostPtr[457]=15; cooValHostPtr[457]=-5.45;
    cooRowIndexHostPtr[458]=115; cooColIndexHostPtr[458]=114; cooValHostPtr[458]=155.0365680897376;
    cooRowIndexHostPtr[459]=115; cooColIndexHostPtr[459]=115; cooValHostPtr[459]=-314.0731361794753;
    cooRowIndexHostPtr[460]=115; cooColIndexHostPtr[460]=116; cooValHostPtr[460]=155.0365680897376;
    cooRowIndexHostPtr[461]=116; cooColIndexHostPtr[461]=16; cooValHostPtr[461]=-5.45;
    cooRowIndexHostPtr[462]=116; cooColIndexHostPtr[462]=115; cooValHostPtr[462]=155.0365680897376;
    cooRowIndexHostPtr[463]=116; cooColIndexHostPtr[463]=116; cooValHostPtr[463]=-314.0731361794753;
    cooRowIndexHostPtr[464]=116; cooColIndexHostPtr[464]=117; cooValHostPtr[464]=155.0365680897376;
    cooRowIndexHostPtr[465]=117; cooColIndexHostPtr[465]=17; cooValHostPtr[465]=-5.45;
    cooRowIndexHostPtr[466]=117; cooColIndexHostPtr[466]=116; cooValHostPtr[466]=155.0365680897376;
    cooRowIndexHostPtr[467]=117; cooColIndexHostPtr[467]=117; cooValHostPtr[467]=-314.0731361794753;
    cooRowIndexHostPtr[468]=117; cooColIndexHostPtr[468]=118; cooValHostPtr[468]=155.0365680897376;
    cooRowIndexHostPtr[469]=118; cooColIndexHostPtr[469]=18; cooValHostPtr[469]=-5.45;
    cooRowIndexHostPtr[470]=118; cooColIndexHostPtr[470]=117; cooValHostPtr[470]=155.0365680897376;
    cooRowIndexHostPtr[471]=118; cooColIndexHostPtr[471]=118; cooValHostPtr[471]=-314.0731361794753;
    cooRowIndexHostPtr[472]=118; cooColIndexHostPtr[472]=119; cooValHostPtr[472]=155.0365680897376;
    cooRowIndexHostPtr[473]=119; cooColIndexHostPtr[473]=19; cooValHostPtr[473]=-5.45;
    cooRowIndexHostPtr[474]=119; cooColIndexHostPtr[474]=118; cooValHostPtr[474]=155.0365680897376;
    cooRowIndexHostPtr[475]=119; cooColIndexHostPtr[475]=119; cooValHostPtr[475]=-314.0731361794753;
    cooRowIndexHostPtr[476]=119; cooColIndexHostPtr[476]=120; cooValHostPtr[476]=155.0365680897376;
    cooRowIndexHostPtr[477]=120; cooColIndexHostPtr[477]=20; cooValHostPtr[477]=-5.45;
    cooRowIndexHostPtr[478]=120; cooColIndexHostPtr[478]=119; cooValHostPtr[478]=155.0365680897376;
    cooRowIndexHostPtr[479]=120; cooColIndexHostPtr[479]=120; cooValHostPtr[479]=-314.0731361794753;
    cooRowIndexHostPtr[480]=120; cooColIndexHostPtr[480]=121; cooValHostPtr[480]=155.0365680897376;
    cooRowIndexHostPtr[481]=121; cooColIndexHostPtr[481]=21; cooValHostPtr[481]=-5.45;
    cooRowIndexHostPtr[482]=121; cooColIndexHostPtr[482]=120; cooValHostPtr[482]=155.0365680897376;
    cooRowIndexHostPtr[483]=121; cooColIndexHostPtr[483]=121; cooValHostPtr[483]=-314.0731361794753;
    cooRowIndexHostPtr[484]=121; cooColIndexHostPtr[484]=122; cooValHostPtr[484]=155.0365680897376;
    cooRowIndexHostPtr[485]=122; cooColIndexHostPtr[485]=22; cooValHostPtr[485]=-5.45;
    cooRowIndexHostPtr[486]=122; cooColIndexHostPtr[486]=121; cooValHostPtr[486]=155.0365680897376;
    cooRowIndexHostPtr[487]=122; cooColIndexHostPtr[487]=122; cooValHostPtr[487]=-314.0731361794753;
    cooRowIndexHostPtr[488]=122; cooColIndexHostPtr[488]=123; cooValHostPtr[488]=155.0365680897376;
    cooRowIndexHostPtr[489]=123; cooColIndexHostPtr[489]=23; cooValHostPtr[489]=-5.45;
    cooRowIndexHostPtr[490]=123; cooColIndexHostPtr[490]=122; cooValHostPtr[490]=155.0365680897376;
    cooRowIndexHostPtr[491]=123; cooColIndexHostPtr[491]=123; cooValHostPtr[491]=-314.0731361794753;
    cooRowIndexHostPtr[492]=123; cooColIndexHostPtr[492]=124; cooValHostPtr[492]=155.0365680897376;
    cooRowIndexHostPtr[493]=124; cooColIndexHostPtr[493]=24; cooValHostPtr[493]=-5.45;
    cooRowIndexHostPtr[494]=124; cooColIndexHostPtr[494]=123; cooValHostPtr[494]=155.0365680897376;
    cooRowIndexHostPtr[495]=124; cooColIndexHostPtr[495]=124; cooValHostPtr[495]=-314.0731361794753;
    cooRowIndexHostPtr[496]=124; cooColIndexHostPtr[496]=125; cooValHostPtr[496]=155.0365680897376;
    cooRowIndexHostPtr[497]=125; cooColIndexHostPtr[497]=25; cooValHostPtr[497]=-5.45;
    cooRowIndexHostPtr[498]=125; cooColIndexHostPtr[498]=124; cooValHostPtr[498]=155.0365680897376;
    cooRowIndexHostPtr[499]=125; cooColIndexHostPtr[499]=125; cooValHostPtr[499]=-314.0731361794753;
    cooRowIndexHostPtr[500]=125; cooColIndexHostPtr[500]=126; cooValHostPtr[500]=155.0365680897376;
    cooRowIndexHostPtr[501]=126; cooColIndexHostPtr[501]=26; cooValHostPtr[501]=-5.45;
    cooRowIndexHostPtr[502]=126; cooColIndexHostPtr[502]=125; cooValHostPtr[502]=155.0365680897376;
    cooRowIndexHostPtr[503]=126; cooColIndexHostPtr[503]=126; cooValHostPtr[503]=-314.0731361794753;
    cooRowIndexHostPtr[504]=126; cooColIndexHostPtr[504]=127; cooValHostPtr[504]=155.0365680897376;
    cooRowIndexHostPtr[505]=127; cooColIndexHostPtr[505]=27; cooValHostPtr[505]=-5.45;
    cooRowIndexHostPtr[506]=127; cooColIndexHostPtr[506]=126; cooValHostPtr[506]=155.0365680897376;
    cooRowIndexHostPtr[507]=127; cooColIndexHostPtr[507]=127; cooValHostPtr[507]=-314.0731361794753;
    cooRowIndexHostPtr[508]=127; cooColIndexHostPtr[508]=128; cooValHostPtr[508]=155.0365680897376;
    cooRowIndexHostPtr[509]=128; cooColIndexHostPtr[509]=28; cooValHostPtr[509]=-5.45;
    cooRowIndexHostPtr[510]=128; cooColIndexHostPtr[510]=127; cooValHostPtr[510]=155.0365680897376;
    cooRowIndexHostPtr[511]=128; cooColIndexHostPtr[511]=128; cooValHostPtr[511]=-314.0731361794753;
    cooRowIndexHostPtr[512]=128; cooColIndexHostPtr[512]=129; cooValHostPtr[512]=155.0365680897376;
    cooRowIndexHostPtr[513]=129; cooColIndexHostPtr[513]=29; cooValHostPtr[513]=-5.45;
    cooRowIndexHostPtr[514]=129; cooColIndexHostPtr[514]=128; cooValHostPtr[514]=155.0365680897376;
    cooRowIndexHostPtr[515]=129; cooColIndexHostPtr[515]=129; cooValHostPtr[515]=-314.0731361794753;
    cooRowIndexHostPtr[516]=129; cooColIndexHostPtr[516]=130; cooValHostPtr[516]=155.0365680897376;
    cooRowIndexHostPtr[517]=130; cooColIndexHostPtr[517]=30; cooValHostPtr[517]=-5.45;
    cooRowIndexHostPtr[518]=130; cooColIndexHostPtr[518]=129; cooValHostPtr[518]=155.0365680897376;
    cooRowIndexHostPtr[519]=130; cooColIndexHostPtr[519]=130; cooValHostPtr[519]=-314.0731361794753;
    cooRowIndexHostPtr[520]=130; cooColIndexHostPtr[520]=131; cooValHostPtr[520]=155.0365680897376;
    cooRowIndexHostPtr[521]=131; cooColIndexHostPtr[521]=31; cooValHostPtr[521]=-5.45;
    cooRowIndexHostPtr[522]=131; cooColIndexHostPtr[522]=130; cooValHostPtr[522]=155.0365680897376;
    cooRowIndexHostPtr[523]=131; cooColIndexHostPtr[523]=131; cooValHostPtr[523]=-314.0731361794753;
    cooRowIndexHostPtr[524]=131; cooColIndexHostPtr[524]=132; cooValHostPtr[524]=155.0365680897376;
    cooRowIndexHostPtr[525]=132; cooColIndexHostPtr[525]=32; cooValHostPtr[525]=-5.45;
    cooRowIndexHostPtr[526]=132; cooColIndexHostPtr[526]=131; cooValHostPtr[526]=155.0365680897376;
    cooRowIndexHostPtr[527]=132; cooColIndexHostPtr[527]=132; cooValHostPtr[527]=-314.0731361794753;
    cooRowIndexHostPtr[528]=132; cooColIndexHostPtr[528]=133; cooValHostPtr[528]=155.0365680897376;
    cooRowIndexHostPtr[529]=133; cooColIndexHostPtr[529]=33; cooValHostPtr[529]=-5.45;
    cooRowIndexHostPtr[530]=133; cooColIndexHostPtr[530]=132; cooValHostPtr[530]=155.0365680897376;
    cooRowIndexHostPtr[531]=133; cooColIndexHostPtr[531]=133; cooValHostPtr[531]=-314.0731361794753;
    cooRowIndexHostPtr[532]=133; cooColIndexHostPtr[532]=134; cooValHostPtr[532]=155.0365680897376;
    cooRowIndexHostPtr[533]=134; cooColIndexHostPtr[533]=34; cooValHostPtr[533]=-5.45;
    cooRowIndexHostPtr[534]=134; cooColIndexHostPtr[534]=133; cooValHostPtr[534]=155.0365680897376;
    cooRowIndexHostPtr[535]=134; cooColIndexHostPtr[535]=134; cooValHostPtr[535]=-314.0731361794753;
    cooRowIndexHostPtr[536]=134; cooColIndexHostPtr[536]=135; cooValHostPtr[536]=155.0365680897376;
    cooRowIndexHostPtr[537]=135; cooColIndexHostPtr[537]=35; cooValHostPtr[537]=-5.45;
    cooRowIndexHostPtr[538]=135; cooColIndexHostPtr[538]=134; cooValHostPtr[538]=155.0365680897376;
    cooRowIndexHostPtr[539]=135; cooColIndexHostPtr[539]=135; cooValHostPtr[539]=-314.0731361794753;
    cooRowIndexHostPtr[540]=135; cooColIndexHostPtr[540]=136; cooValHostPtr[540]=155.0365680897376;
    cooRowIndexHostPtr[541]=136; cooColIndexHostPtr[541]=36; cooValHostPtr[541]=-5.45;
    cooRowIndexHostPtr[542]=136; cooColIndexHostPtr[542]=135; cooValHostPtr[542]=155.0365680897376;
    cooRowIndexHostPtr[543]=136; cooColIndexHostPtr[543]=136; cooValHostPtr[543]=-314.0731361794753;
    cooRowIndexHostPtr[544]=136; cooColIndexHostPtr[544]=137; cooValHostPtr[544]=155.0365680897376;
    cooRowIndexHostPtr[545]=137; cooColIndexHostPtr[545]=37; cooValHostPtr[545]=-5.45;
    cooRowIndexHostPtr[546]=137; cooColIndexHostPtr[546]=136; cooValHostPtr[546]=155.0365680897376;
    cooRowIndexHostPtr[547]=137; cooColIndexHostPtr[547]=137; cooValHostPtr[547]=-314.0731361794753;
    cooRowIndexHostPtr[548]=137; cooColIndexHostPtr[548]=138; cooValHostPtr[548]=155.0365680897376;
    cooRowIndexHostPtr[549]=138; cooColIndexHostPtr[549]=38; cooValHostPtr[549]=-5.45;
    cooRowIndexHostPtr[550]=138; cooColIndexHostPtr[550]=137; cooValHostPtr[550]=155.0365680897376;
    cooRowIndexHostPtr[551]=138; cooColIndexHostPtr[551]=138; cooValHostPtr[551]=-314.0731361794753;
    cooRowIndexHostPtr[552]=138; cooColIndexHostPtr[552]=139; cooValHostPtr[552]=155.0365680897376;
    cooRowIndexHostPtr[553]=139; cooColIndexHostPtr[553]=39; cooValHostPtr[553]=-5.45;
    cooRowIndexHostPtr[554]=139; cooColIndexHostPtr[554]=138; cooValHostPtr[554]=155.0365680897376;
    cooRowIndexHostPtr[555]=139; cooColIndexHostPtr[555]=139; cooValHostPtr[555]=-314.0731361794753;
    cooRowIndexHostPtr[556]=139; cooColIndexHostPtr[556]=140; cooValHostPtr[556]=155.0365680897376;
    cooRowIndexHostPtr[557]=140; cooColIndexHostPtr[557]=40; cooValHostPtr[557]=-5.45;
    cooRowIndexHostPtr[558]=140; cooColIndexHostPtr[558]=139; cooValHostPtr[558]=155.0365680897376;
    cooRowIndexHostPtr[559]=140; cooColIndexHostPtr[559]=140; cooValHostPtr[559]=-314.0731361794753;
    cooRowIndexHostPtr[560]=140; cooColIndexHostPtr[560]=141; cooValHostPtr[560]=155.0365680897376;
    cooRowIndexHostPtr[561]=141; cooColIndexHostPtr[561]=41; cooValHostPtr[561]=-5.45;
    cooRowIndexHostPtr[562]=141; cooColIndexHostPtr[562]=140; cooValHostPtr[562]=155.0365680897376;
    cooRowIndexHostPtr[563]=141; cooColIndexHostPtr[563]=141; cooValHostPtr[563]=-314.0731361794753;
    cooRowIndexHostPtr[564]=141; cooColIndexHostPtr[564]=142; cooValHostPtr[564]=155.0365680897376;
    cooRowIndexHostPtr[565]=142; cooColIndexHostPtr[565]=42; cooValHostPtr[565]=-5.45;
    cooRowIndexHostPtr[566]=142; cooColIndexHostPtr[566]=141; cooValHostPtr[566]=155.0365680897376;
    cooRowIndexHostPtr[567]=142; cooColIndexHostPtr[567]=142; cooValHostPtr[567]=-314.0731361794753;
    cooRowIndexHostPtr[568]=142; cooColIndexHostPtr[568]=143; cooValHostPtr[568]=155.0365680897376;
    cooRowIndexHostPtr[569]=143; cooColIndexHostPtr[569]=43; cooValHostPtr[569]=-5.45;
    cooRowIndexHostPtr[570]=143; cooColIndexHostPtr[570]=142; cooValHostPtr[570]=155.0365680897376;
    cooRowIndexHostPtr[571]=143; cooColIndexHostPtr[571]=143; cooValHostPtr[571]=-314.0731361794753;
    cooRowIndexHostPtr[572]=143; cooColIndexHostPtr[572]=144; cooValHostPtr[572]=155.0365680897376;
    cooRowIndexHostPtr[573]=144; cooColIndexHostPtr[573]=44; cooValHostPtr[573]=-5.45;
    cooRowIndexHostPtr[574]=144; cooColIndexHostPtr[574]=143; cooValHostPtr[574]=155.0365680897376;
    cooRowIndexHostPtr[575]=144; cooColIndexHostPtr[575]=144; cooValHostPtr[575]=-314.0731361794753;
    cooRowIndexHostPtr[576]=144; cooColIndexHostPtr[576]=145; cooValHostPtr[576]=155.0365680897376;
    cooRowIndexHostPtr[577]=145; cooColIndexHostPtr[577]=45; cooValHostPtr[577]=-5.45;
    cooRowIndexHostPtr[578]=145; cooColIndexHostPtr[578]=144; cooValHostPtr[578]=155.0365680897376;
    cooRowIndexHostPtr[579]=145; cooColIndexHostPtr[579]=145; cooValHostPtr[579]=-314.0731361794753;
    cooRowIndexHostPtr[580]=145; cooColIndexHostPtr[580]=146; cooValHostPtr[580]=155.0365680897376;
    cooRowIndexHostPtr[581]=146; cooColIndexHostPtr[581]=46; cooValHostPtr[581]=-5.45;
    cooRowIndexHostPtr[582]=146; cooColIndexHostPtr[582]=145; cooValHostPtr[582]=155.0365680897376;
    cooRowIndexHostPtr[583]=146; cooColIndexHostPtr[583]=146; cooValHostPtr[583]=-314.0731361794753;
    cooRowIndexHostPtr[584]=146; cooColIndexHostPtr[584]=147; cooValHostPtr[584]=155.0365680897376;
    cooRowIndexHostPtr[585]=147; cooColIndexHostPtr[585]=47; cooValHostPtr[585]=-5.45;
    cooRowIndexHostPtr[586]=147; cooColIndexHostPtr[586]=146; cooValHostPtr[586]=155.0365680897376;
    cooRowIndexHostPtr[587]=147; cooColIndexHostPtr[587]=147; cooValHostPtr[587]=-314.0731361794753;
    cooRowIndexHostPtr[588]=147; cooColIndexHostPtr[588]=148; cooValHostPtr[588]=155.0365680897376;
    cooRowIndexHostPtr[589]=148; cooColIndexHostPtr[589]=48; cooValHostPtr[589]=-5.45;
    cooRowIndexHostPtr[590]=148; cooColIndexHostPtr[590]=147; cooValHostPtr[590]=155.0365680897376;
    cooRowIndexHostPtr[591]=148; cooColIndexHostPtr[591]=148; cooValHostPtr[591]=-314.0731361794753;
    cooRowIndexHostPtr[592]=148; cooColIndexHostPtr[592]=149; cooValHostPtr[592]=155.0365680897376;
    cooRowIndexHostPtr[593]=149; cooColIndexHostPtr[593]=49; cooValHostPtr[593]=-5.45;
    cooRowIndexHostPtr[594]=149; cooColIndexHostPtr[594]=148; cooValHostPtr[594]=155.0365680897376;
    cooRowIndexHostPtr[595]=149; cooColIndexHostPtr[595]=149; cooValHostPtr[595]=-314.0731361794753;
    cooRowIndexHostPtr[596]=149; cooColIndexHostPtr[596]=150; cooValHostPtr[596]=155.0365680897376;
    cooRowIndexHostPtr[597]=150; cooColIndexHostPtr[597]=50; cooValHostPtr[597]=-5.45;
    cooRowIndexHostPtr[598]=150; cooColIndexHostPtr[598]=149; cooValHostPtr[598]=155.0365680897376;
    cooRowIndexHostPtr[599]=150; cooColIndexHostPtr[599]=150; cooValHostPtr[599]=-314.0731361794753;
    cooRowIndexHostPtr[600]=150; cooColIndexHostPtr[600]=151; cooValHostPtr[600]=155.0365680897376;
    cooRowIndexHostPtr[601]=151; cooColIndexHostPtr[601]=51; cooValHostPtr[601]=-5.45;
    cooRowIndexHostPtr[602]=151; cooColIndexHostPtr[602]=150; cooValHostPtr[602]=155.0365680897376;
    cooRowIndexHostPtr[603]=151; cooColIndexHostPtr[603]=151; cooValHostPtr[603]=-314.0731361794753;
    cooRowIndexHostPtr[604]=151; cooColIndexHostPtr[604]=152; cooValHostPtr[604]=155.0365680897376;
    cooRowIndexHostPtr[605]=152; cooColIndexHostPtr[605]=52; cooValHostPtr[605]=-5.45;
    cooRowIndexHostPtr[606]=152; cooColIndexHostPtr[606]=151; cooValHostPtr[606]=155.0365680897376;
    cooRowIndexHostPtr[607]=152; cooColIndexHostPtr[607]=152; cooValHostPtr[607]=-314.0731361794753;
    cooRowIndexHostPtr[608]=152; cooColIndexHostPtr[608]=153; cooValHostPtr[608]=155.0365680897376;
    cooRowIndexHostPtr[609]=153; cooColIndexHostPtr[609]=53; cooValHostPtr[609]=-5.45;
    cooRowIndexHostPtr[610]=153; cooColIndexHostPtr[610]=152; cooValHostPtr[610]=155.0365680897376;
    cooRowIndexHostPtr[611]=153; cooColIndexHostPtr[611]=153; cooValHostPtr[611]=-314.0731361794753;
    cooRowIndexHostPtr[612]=153; cooColIndexHostPtr[612]=154; cooValHostPtr[612]=155.0365680897376;
    cooRowIndexHostPtr[613]=154; cooColIndexHostPtr[613]=54; cooValHostPtr[613]=-5.45;
    cooRowIndexHostPtr[614]=154; cooColIndexHostPtr[614]=153; cooValHostPtr[614]=155.0365680897376;
    cooRowIndexHostPtr[615]=154; cooColIndexHostPtr[615]=154; cooValHostPtr[615]=-314.0731361794753;
    cooRowIndexHostPtr[616]=154; cooColIndexHostPtr[616]=155; cooValHostPtr[616]=155.0365680897376;
    cooRowIndexHostPtr[617]=155; cooColIndexHostPtr[617]=55; cooValHostPtr[617]=-5.45;
    cooRowIndexHostPtr[618]=155; cooColIndexHostPtr[618]=154; cooValHostPtr[618]=155.0365680897376;
    cooRowIndexHostPtr[619]=155; cooColIndexHostPtr[619]=155; cooValHostPtr[619]=-314.0731361794753;
    cooRowIndexHostPtr[620]=155; cooColIndexHostPtr[620]=156; cooValHostPtr[620]=155.0365680897376;
    cooRowIndexHostPtr[621]=156; cooColIndexHostPtr[621]=56; cooValHostPtr[621]=-5.45;
    cooRowIndexHostPtr[622]=156; cooColIndexHostPtr[622]=155; cooValHostPtr[622]=155.0365680897376;
    cooRowIndexHostPtr[623]=156; cooColIndexHostPtr[623]=156; cooValHostPtr[623]=-314.0731361794753;
    cooRowIndexHostPtr[624]=156; cooColIndexHostPtr[624]=157; cooValHostPtr[624]=155.0365680897376;
    cooRowIndexHostPtr[625]=157; cooColIndexHostPtr[625]=57; cooValHostPtr[625]=-5.45;
    cooRowIndexHostPtr[626]=157; cooColIndexHostPtr[626]=156; cooValHostPtr[626]=155.0365680897376;
    cooRowIndexHostPtr[627]=157; cooColIndexHostPtr[627]=157; cooValHostPtr[627]=-314.0731361794753;
    cooRowIndexHostPtr[628]=157; cooColIndexHostPtr[628]=158; cooValHostPtr[628]=155.0365680897376;
    cooRowIndexHostPtr[629]=158; cooColIndexHostPtr[629]=58; cooValHostPtr[629]=-5.45;
    cooRowIndexHostPtr[630]=158; cooColIndexHostPtr[630]=157; cooValHostPtr[630]=155.0365680897376;
    cooRowIndexHostPtr[631]=158; cooColIndexHostPtr[631]=158; cooValHostPtr[631]=-314.0731361794753;
    cooRowIndexHostPtr[632]=158; cooColIndexHostPtr[632]=159; cooValHostPtr[632]=155.0365680897376;
    cooRowIndexHostPtr[633]=159; cooColIndexHostPtr[633]=59; cooValHostPtr[633]=-5.45;
    cooRowIndexHostPtr[634]=159; cooColIndexHostPtr[634]=158; cooValHostPtr[634]=155.0365680897376;
    cooRowIndexHostPtr[635]=159; cooColIndexHostPtr[635]=159; cooValHostPtr[635]=-314.0731361794753;
    cooRowIndexHostPtr[636]=159; cooColIndexHostPtr[636]=160; cooValHostPtr[636]=155.0365680897376;
    cooRowIndexHostPtr[637]=160; cooColIndexHostPtr[637]=60; cooValHostPtr[637]=-5.45;
    cooRowIndexHostPtr[638]=160; cooColIndexHostPtr[638]=159; cooValHostPtr[638]=155.0365680897376;
    cooRowIndexHostPtr[639]=160; cooColIndexHostPtr[639]=160; cooValHostPtr[639]=-314.0731361794753;
    cooRowIndexHostPtr[640]=160; cooColIndexHostPtr[640]=161; cooValHostPtr[640]=155.0365680897376;
    cooRowIndexHostPtr[641]=161; cooColIndexHostPtr[641]=61; cooValHostPtr[641]=-5.45;
    cooRowIndexHostPtr[642]=161; cooColIndexHostPtr[642]=160; cooValHostPtr[642]=155.0365680897376;
    cooRowIndexHostPtr[643]=161; cooColIndexHostPtr[643]=161; cooValHostPtr[643]=-314.0731361794753;
    cooRowIndexHostPtr[644]=161; cooColIndexHostPtr[644]=162; cooValHostPtr[644]=155.0365680897376;
    cooRowIndexHostPtr[645]=162; cooColIndexHostPtr[645]=62; cooValHostPtr[645]=-5.45;
    cooRowIndexHostPtr[646]=162; cooColIndexHostPtr[646]=161; cooValHostPtr[646]=155.0365680897376;
    cooRowIndexHostPtr[647]=162; cooColIndexHostPtr[647]=162; cooValHostPtr[647]=-314.0731361794753;
    cooRowIndexHostPtr[648]=162; cooColIndexHostPtr[648]=163; cooValHostPtr[648]=155.0365680897376;
    cooRowIndexHostPtr[649]=163; cooColIndexHostPtr[649]=63; cooValHostPtr[649]=-5.45;
    cooRowIndexHostPtr[650]=163; cooColIndexHostPtr[650]=162; cooValHostPtr[650]=155.0365680897376;
    cooRowIndexHostPtr[651]=163; cooColIndexHostPtr[651]=163; cooValHostPtr[651]=-314.0731361794753;
    cooRowIndexHostPtr[652]=163; cooColIndexHostPtr[652]=164; cooValHostPtr[652]=155.0365680897376;
    cooRowIndexHostPtr[653]=164; cooColIndexHostPtr[653]=64; cooValHostPtr[653]=-5.45;
    cooRowIndexHostPtr[654]=164; cooColIndexHostPtr[654]=163; cooValHostPtr[654]=155.0365680897376;
    cooRowIndexHostPtr[655]=164; cooColIndexHostPtr[655]=164; cooValHostPtr[655]=-314.0731361794753;
    cooRowIndexHostPtr[656]=164; cooColIndexHostPtr[656]=165; cooValHostPtr[656]=155.0365680897376;
    cooRowIndexHostPtr[657]=165; cooColIndexHostPtr[657]=65; cooValHostPtr[657]=-5.45;
    cooRowIndexHostPtr[658]=165; cooColIndexHostPtr[658]=164; cooValHostPtr[658]=155.0365680897376;
    cooRowIndexHostPtr[659]=165; cooColIndexHostPtr[659]=165; cooValHostPtr[659]=-314.0731361794753;
    cooRowIndexHostPtr[660]=165; cooColIndexHostPtr[660]=166; cooValHostPtr[660]=155.0365680897376;
    cooRowIndexHostPtr[661]=166; cooColIndexHostPtr[661]=66; cooValHostPtr[661]=-5.45;
    cooRowIndexHostPtr[662]=166; cooColIndexHostPtr[662]=165; cooValHostPtr[662]=155.0365680897376;
    cooRowIndexHostPtr[663]=166; cooColIndexHostPtr[663]=166; cooValHostPtr[663]=-314.0731361794753;
    cooRowIndexHostPtr[664]=166; cooColIndexHostPtr[664]=167; cooValHostPtr[664]=155.0365680897376;
    cooRowIndexHostPtr[665]=167; cooColIndexHostPtr[665]=67; cooValHostPtr[665]=-5.45;
    cooRowIndexHostPtr[666]=167; cooColIndexHostPtr[666]=166; cooValHostPtr[666]=155.0365680897376;
    cooRowIndexHostPtr[667]=167; cooColIndexHostPtr[667]=167; cooValHostPtr[667]=-314.0731361794753;
    cooRowIndexHostPtr[668]=167; cooColIndexHostPtr[668]=168; cooValHostPtr[668]=155.0365680897376;
    cooRowIndexHostPtr[669]=168; cooColIndexHostPtr[669]=68; cooValHostPtr[669]=-5.45;
    cooRowIndexHostPtr[670]=168; cooColIndexHostPtr[670]=167; cooValHostPtr[670]=155.0365680897376;
    cooRowIndexHostPtr[671]=168; cooColIndexHostPtr[671]=168; cooValHostPtr[671]=-314.0731361794753;
    cooRowIndexHostPtr[672]=168; cooColIndexHostPtr[672]=169; cooValHostPtr[672]=155.0365680897376;
    cooRowIndexHostPtr[673]=169; cooColIndexHostPtr[673]=69; cooValHostPtr[673]=-5.45;
    cooRowIndexHostPtr[674]=169; cooColIndexHostPtr[674]=168; cooValHostPtr[674]=155.0365680897376;
    cooRowIndexHostPtr[675]=169; cooColIndexHostPtr[675]=169; cooValHostPtr[675]=-314.0731361794753;
    cooRowIndexHostPtr[676]=169; cooColIndexHostPtr[676]=170; cooValHostPtr[676]=155.0365680897376;
    cooRowIndexHostPtr[677]=170; cooColIndexHostPtr[677]=70; cooValHostPtr[677]=-5.45;
    cooRowIndexHostPtr[678]=170; cooColIndexHostPtr[678]=169; cooValHostPtr[678]=155.0365680897376;
    cooRowIndexHostPtr[679]=170; cooColIndexHostPtr[679]=170; cooValHostPtr[679]=-314.0731361794753;
    cooRowIndexHostPtr[680]=170; cooColIndexHostPtr[680]=171; cooValHostPtr[680]=155.0365680897376;
    cooRowIndexHostPtr[681]=171; cooColIndexHostPtr[681]=71; cooValHostPtr[681]=-5.45;
    cooRowIndexHostPtr[682]=171; cooColIndexHostPtr[682]=170; cooValHostPtr[682]=155.0365680897376;
    cooRowIndexHostPtr[683]=171; cooColIndexHostPtr[683]=171; cooValHostPtr[683]=-314.0731361794753;
    cooRowIndexHostPtr[684]=171; cooColIndexHostPtr[684]=172; cooValHostPtr[684]=155.0365680897376;
    cooRowIndexHostPtr[685]=172; cooColIndexHostPtr[685]=72; cooValHostPtr[685]=-5.45;
    cooRowIndexHostPtr[686]=172; cooColIndexHostPtr[686]=171; cooValHostPtr[686]=155.0365680897376;
    cooRowIndexHostPtr[687]=172; cooColIndexHostPtr[687]=172; cooValHostPtr[687]=-314.0731361794753;
    cooRowIndexHostPtr[688]=172; cooColIndexHostPtr[688]=173; cooValHostPtr[688]=155.0365680897376;
    cooRowIndexHostPtr[689]=173; cooColIndexHostPtr[689]=73; cooValHostPtr[689]=-5.45;
    cooRowIndexHostPtr[690]=173; cooColIndexHostPtr[690]=172; cooValHostPtr[690]=155.0365680897376;
    cooRowIndexHostPtr[691]=173; cooColIndexHostPtr[691]=173; cooValHostPtr[691]=-314.0731361794753;
    cooRowIndexHostPtr[692]=173; cooColIndexHostPtr[692]=174; cooValHostPtr[692]=155.0365680897376;
    cooRowIndexHostPtr[693]=174; cooColIndexHostPtr[693]=74; cooValHostPtr[693]=-5.45;
    cooRowIndexHostPtr[694]=174; cooColIndexHostPtr[694]=173; cooValHostPtr[694]=155.0365680897376;
    cooRowIndexHostPtr[695]=174; cooColIndexHostPtr[695]=174; cooValHostPtr[695]=-314.0731361794753;
    cooRowIndexHostPtr[696]=174; cooColIndexHostPtr[696]=175; cooValHostPtr[696]=155.0365680897376;
    cooRowIndexHostPtr[697]=175; cooColIndexHostPtr[697]=75; cooValHostPtr[697]=-5.45;
    cooRowIndexHostPtr[698]=175; cooColIndexHostPtr[698]=174; cooValHostPtr[698]=155.0365680897376;
    cooRowIndexHostPtr[699]=175; cooColIndexHostPtr[699]=175; cooValHostPtr[699]=-314.0731361794753;
    cooRowIndexHostPtr[700]=175; cooColIndexHostPtr[700]=176; cooValHostPtr[700]=155.0365680897376;
    cooRowIndexHostPtr[701]=176; cooColIndexHostPtr[701]=76; cooValHostPtr[701]=-5.45;
    cooRowIndexHostPtr[702]=176; cooColIndexHostPtr[702]=175; cooValHostPtr[702]=155.0365680897376;
    cooRowIndexHostPtr[703]=176; cooColIndexHostPtr[703]=176; cooValHostPtr[703]=-314.0731361794753;
    cooRowIndexHostPtr[704]=176; cooColIndexHostPtr[704]=177; cooValHostPtr[704]=155.0365680897376;
    cooRowIndexHostPtr[705]=177; cooColIndexHostPtr[705]=77; cooValHostPtr[705]=-5.45;
    cooRowIndexHostPtr[706]=177; cooColIndexHostPtr[706]=176; cooValHostPtr[706]=155.0365680897376;
    cooRowIndexHostPtr[707]=177; cooColIndexHostPtr[707]=177; cooValHostPtr[707]=-314.0731361794753;
    cooRowIndexHostPtr[708]=177; cooColIndexHostPtr[708]=178; cooValHostPtr[708]=155.0365680897376;
    cooRowIndexHostPtr[709]=178; cooColIndexHostPtr[709]=78; cooValHostPtr[709]=-5.45;
    cooRowIndexHostPtr[710]=178; cooColIndexHostPtr[710]=177; cooValHostPtr[710]=155.0365680897376;
    cooRowIndexHostPtr[711]=178; cooColIndexHostPtr[711]=178; cooValHostPtr[711]=-314.0731361794753;
    cooRowIndexHostPtr[712]=178; cooColIndexHostPtr[712]=179; cooValHostPtr[712]=155.0365680897376;
    cooRowIndexHostPtr[713]=179; cooColIndexHostPtr[713]=79; cooValHostPtr[713]=-5.45;
    cooRowIndexHostPtr[714]=179; cooColIndexHostPtr[714]=178; cooValHostPtr[714]=155.0365680897376;
    cooRowIndexHostPtr[715]=179; cooColIndexHostPtr[715]=179; cooValHostPtr[715]=-314.0731361794753;
    cooRowIndexHostPtr[716]=179; cooColIndexHostPtr[716]=180; cooValHostPtr[716]=155.0365680897376;
    cooRowIndexHostPtr[717]=180; cooColIndexHostPtr[717]=80; cooValHostPtr[717]=-5.45;
    cooRowIndexHostPtr[718]=180; cooColIndexHostPtr[718]=179; cooValHostPtr[718]=155.0365680897376;
    cooRowIndexHostPtr[719]=180; cooColIndexHostPtr[719]=180; cooValHostPtr[719]=-314.0731361794753;
    cooRowIndexHostPtr[720]=180; cooColIndexHostPtr[720]=181; cooValHostPtr[720]=155.0365680897376;
    cooRowIndexHostPtr[721]=181; cooColIndexHostPtr[721]=81; cooValHostPtr[721]=-5.45;
    cooRowIndexHostPtr[722]=181; cooColIndexHostPtr[722]=180; cooValHostPtr[722]=155.0365680897376;
    cooRowIndexHostPtr[723]=181; cooColIndexHostPtr[723]=181; cooValHostPtr[723]=-314.0731361794753;
    cooRowIndexHostPtr[724]=181; cooColIndexHostPtr[724]=182; cooValHostPtr[724]=155.0365680897376;
    cooRowIndexHostPtr[725]=182; cooColIndexHostPtr[725]=82; cooValHostPtr[725]=-5.45;
    cooRowIndexHostPtr[726]=182; cooColIndexHostPtr[726]=181; cooValHostPtr[726]=155.0365680897376;
    cooRowIndexHostPtr[727]=182; cooColIndexHostPtr[727]=182; cooValHostPtr[727]=-314.0731361794753;
    cooRowIndexHostPtr[728]=182; cooColIndexHostPtr[728]=183; cooValHostPtr[728]=155.0365680897376;
    cooRowIndexHostPtr[729]=183; cooColIndexHostPtr[729]=83; cooValHostPtr[729]=-5.45;
    cooRowIndexHostPtr[730]=183; cooColIndexHostPtr[730]=182; cooValHostPtr[730]=155.0365680897376;
    cooRowIndexHostPtr[731]=183; cooColIndexHostPtr[731]=183; cooValHostPtr[731]=-314.0731361794753;
    cooRowIndexHostPtr[732]=183; cooColIndexHostPtr[732]=184; cooValHostPtr[732]=155.0365680897376;
    cooRowIndexHostPtr[733]=184; cooColIndexHostPtr[733]=84; cooValHostPtr[733]=-5.45;
    cooRowIndexHostPtr[734]=184; cooColIndexHostPtr[734]=183; cooValHostPtr[734]=155.0365680897376;
    cooRowIndexHostPtr[735]=184; cooColIndexHostPtr[735]=184; cooValHostPtr[735]=-314.0731361794753;
    cooRowIndexHostPtr[736]=184; cooColIndexHostPtr[736]=185; cooValHostPtr[736]=155.0365680897376;
    cooRowIndexHostPtr[737]=185; cooColIndexHostPtr[737]=85; cooValHostPtr[737]=-5.45;
    cooRowIndexHostPtr[738]=185; cooColIndexHostPtr[738]=184; cooValHostPtr[738]=155.0365680897376;
    cooRowIndexHostPtr[739]=185; cooColIndexHostPtr[739]=185; cooValHostPtr[739]=-314.0731361794753;
    cooRowIndexHostPtr[740]=185; cooColIndexHostPtr[740]=186; cooValHostPtr[740]=155.0365680897376;
    cooRowIndexHostPtr[741]=186; cooColIndexHostPtr[741]=86; cooValHostPtr[741]=-5.45;
    cooRowIndexHostPtr[742]=186; cooColIndexHostPtr[742]=185; cooValHostPtr[742]=155.0365680897376;
    cooRowIndexHostPtr[743]=186; cooColIndexHostPtr[743]=186; cooValHostPtr[743]=-314.0731361794753;
    cooRowIndexHostPtr[744]=186; cooColIndexHostPtr[744]=187; cooValHostPtr[744]=155.0365680897376;
    cooRowIndexHostPtr[745]=187; cooColIndexHostPtr[745]=87; cooValHostPtr[745]=-5.45;
    cooRowIndexHostPtr[746]=187; cooColIndexHostPtr[746]=186; cooValHostPtr[746]=155.0365680897376;
    cooRowIndexHostPtr[747]=187; cooColIndexHostPtr[747]=187; cooValHostPtr[747]=-314.0731361794753;
    cooRowIndexHostPtr[748]=187; cooColIndexHostPtr[748]=188; cooValHostPtr[748]=155.0365680897376;
    cooRowIndexHostPtr[749]=188; cooColIndexHostPtr[749]=88; cooValHostPtr[749]=-5.45;
    cooRowIndexHostPtr[750]=188; cooColIndexHostPtr[750]=187; cooValHostPtr[750]=155.0365680897376;
    cooRowIndexHostPtr[751]=188; cooColIndexHostPtr[751]=188; cooValHostPtr[751]=-314.0731361794753;
    cooRowIndexHostPtr[752]=188; cooColIndexHostPtr[752]=189; cooValHostPtr[752]=155.0365680897376;
    cooRowIndexHostPtr[753]=189; cooColIndexHostPtr[753]=89; cooValHostPtr[753]=-5.45;
    cooRowIndexHostPtr[754]=189; cooColIndexHostPtr[754]=188; cooValHostPtr[754]=155.0365680897376;
    cooRowIndexHostPtr[755]=189; cooColIndexHostPtr[755]=189; cooValHostPtr[755]=-314.0731361794753;
    cooRowIndexHostPtr[756]=189; cooColIndexHostPtr[756]=190; cooValHostPtr[756]=155.0365680897376;
    cooRowIndexHostPtr[757]=190; cooColIndexHostPtr[757]=90; cooValHostPtr[757]=-5.45;
    cooRowIndexHostPtr[758]=190; cooColIndexHostPtr[758]=189; cooValHostPtr[758]=155.0365680897376;
    cooRowIndexHostPtr[759]=190; cooColIndexHostPtr[759]=190; cooValHostPtr[759]=-314.0731361794753;
    cooRowIndexHostPtr[760]=190; cooColIndexHostPtr[760]=191; cooValHostPtr[760]=155.0365680897376;
    cooRowIndexHostPtr[761]=191; cooColIndexHostPtr[761]=91; cooValHostPtr[761]=-5.45;
    cooRowIndexHostPtr[762]=191; cooColIndexHostPtr[762]=190; cooValHostPtr[762]=155.0365680897376;
    cooRowIndexHostPtr[763]=191; cooColIndexHostPtr[763]=191; cooValHostPtr[763]=-314.0731361794753;
    cooRowIndexHostPtr[764]=191; cooColIndexHostPtr[764]=192; cooValHostPtr[764]=155.0365680897376;
    cooRowIndexHostPtr[765]=192; cooColIndexHostPtr[765]=92; cooValHostPtr[765]=-5.45;
    cooRowIndexHostPtr[766]=192; cooColIndexHostPtr[766]=191; cooValHostPtr[766]=155.0365680897376;
    cooRowIndexHostPtr[767]=192; cooColIndexHostPtr[767]=192; cooValHostPtr[767]=-314.0731361794753;
    cooRowIndexHostPtr[768]=192; cooColIndexHostPtr[768]=193; cooValHostPtr[768]=155.0365680897376;
    cooRowIndexHostPtr[769]=193; cooColIndexHostPtr[769]=93; cooValHostPtr[769]=-5.45;
    cooRowIndexHostPtr[770]=193; cooColIndexHostPtr[770]=192; cooValHostPtr[770]=155.0365680897376;
    cooRowIndexHostPtr[771]=193; cooColIndexHostPtr[771]=193; cooValHostPtr[771]=-314.0731361794753;
    cooRowIndexHostPtr[772]=193; cooColIndexHostPtr[772]=194; cooValHostPtr[772]=155.0365680897376;
    cooRowIndexHostPtr[773]=194; cooColIndexHostPtr[773]=94; cooValHostPtr[773]=-5.45;
    cooRowIndexHostPtr[774]=194; cooColIndexHostPtr[774]=193; cooValHostPtr[774]=155.0365680897376;
    cooRowIndexHostPtr[775]=194; cooColIndexHostPtr[775]=194; cooValHostPtr[775]=-314.0731361794753;
    cooRowIndexHostPtr[776]=194; cooColIndexHostPtr[776]=195; cooValHostPtr[776]=155.0365680897376;
    cooRowIndexHostPtr[777]=195; cooColIndexHostPtr[777]=95; cooValHostPtr[777]=-5.45;
    cooRowIndexHostPtr[778]=195; cooColIndexHostPtr[778]=194; cooValHostPtr[778]=155.0365680897376;
    cooRowIndexHostPtr[779]=195; cooColIndexHostPtr[779]=195; cooValHostPtr[779]=-314.0731361794753;
    cooRowIndexHostPtr[780]=195; cooColIndexHostPtr[780]=196; cooValHostPtr[780]=155.0365680897376;
    cooRowIndexHostPtr[781]=196; cooColIndexHostPtr[781]=96; cooValHostPtr[781]=-5.45;
    cooRowIndexHostPtr[782]=196; cooColIndexHostPtr[782]=195; cooValHostPtr[782]=155.0365680897376;
    cooRowIndexHostPtr[783]=196; cooColIndexHostPtr[783]=196; cooValHostPtr[783]=-314.0731361794753;
    cooRowIndexHostPtr[784]=196; cooColIndexHostPtr[784]=197; cooValHostPtr[784]=155.0365680897376;
    cooRowIndexHostPtr[785]=197; cooColIndexHostPtr[785]=97; cooValHostPtr[785]=-5.45;
    cooRowIndexHostPtr[786]=197; cooColIndexHostPtr[786]=196; cooValHostPtr[786]=155.0365680897376;
    cooRowIndexHostPtr[787]=197; cooColIndexHostPtr[787]=197; cooValHostPtr[787]=-314.0731361794753;
    cooRowIndexHostPtr[788]=197; cooColIndexHostPtr[788]=198; cooValHostPtr[788]=155.0365680897376;
    cooRowIndexHostPtr[789]=198; cooColIndexHostPtr[789]=98; cooValHostPtr[789]=-5.45;
    cooRowIndexHostPtr[790]=198; cooColIndexHostPtr[790]=197; cooValHostPtr[790]=155.0365680897376;
    cooRowIndexHostPtr[791]=198; cooColIndexHostPtr[791]=198; cooValHostPtr[791]=-314.0731361794753;
    cooRowIndexHostPtr[792]=198; cooColIndexHostPtr[792]=199; cooValHostPtr[792]=155.0365680897376;
    cooRowIndexHostPtr[793]=199; cooColIndexHostPtr[793]=99; cooValHostPtr[793]=-5.45;
    cooRowIndexHostPtr[794]=199; cooColIndexHostPtr[794]=198; cooValHostPtr[794]=155.0365680897376;
    cooRowIndexHostPtr[795]=199; cooColIndexHostPtr[795]=199; cooValHostPtr[795]=-314.0731361794753;
    
    /*
    //print the matrix
    printf("Input data:\n");
    for (int i=0; i<nnz; i++){
        printf("cooRowIndexHostPtr[%d]=%d  ",i,cooRowIndexHostPtr[i]);
        printf("cooColIndexHostPtr[%d]=%d  ",i,cooColIndexHostPtr[i]);
        printf("cooValHostPtr[%d]=%f     \n",i,cooValHostPtr[i]);
    }
    */

    /* create a dense vector */
    /*  y  = [1.0 2.0 3.0 4.0 5.0] (dense) */
    yHostPtr    = (double *)malloc(n       *sizeof(yHostPtr[0]));
    y_static    = (double *)malloc(n       *sizeof(yHostPtr[0]));
    if(!yHostPtr || !y_static){
        CLEANUP("Host malloc failed (vectors)");
        return 1;
    }

    srand (time(NULL));
    for(int i = 0; i < n; i++){
        y_static[i] = rand() / double(RAND_MAX);
    }

    /*
    //print the vectors
    for (int j=0; j<1; j++){
        for (int i=0; i<n; i++){
            printf("yHostPtr[%d,%d]=%f\n",i,j,yHostPtr[i+n*j]);
        }
    }
    */

    /* allocate GPU memory and copy the matrix and vectors into it */
    cudaStat1 = hipMalloc((void**)&cooRowIndex,nnz*sizeof(cooRowIndex[0]));
    cudaStat2 = hipMalloc((void**)&cooColIndex,nnz*sizeof(cooColIndex[0]));
    cudaStat3 = hipMalloc((void**)&cooVal,     nnz*sizeof(cooVal[0]));
    cudaStat4 = hipMalloc((void**)&y,          n*sizeof(y[0]));
    if ((cudaStat1 != hipSuccess) ||
        (cudaStat2 != hipSuccess) ||
        (cudaStat3 != hipSuccess) ||
        (cudaStat4 != hipSuccess)) {
        CLEANUP("Device malloc failed");
        return 1;
    }
    cudaStat1 = hipMemcpy(cooRowIndex, cooRowIndexHostPtr,
                           (size_t)(nnz*sizeof(cooRowIndex[0])),
                           hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(cooColIndex, cooColIndexHostPtr,
                           (size_t)(nnz*sizeof(cooColIndex[0])),
                           hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(cooVal,      cooValHostPtr,
                           (size_t)(nnz*sizeof(cooVal[0])),
                           hipMemcpyHostToDevice);
    cudaStat4 = hipMemcpy(y,           y_static,
                           (size_t)(n*sizeof(y[0])),
                           hipMemcpyHostToDevice);
    if ((cudaStat1 != hipSuccess) ||
        (cudaStat2 != hipSuccess) ||
        (cudaStat3 != hipSuccess) ||
        (cudaStat4 != hipSuccess)) {
        CLEANUP("Memcpy from Host to Device failed");
        return 1;
    }

    /* initialize cusparse library */
    status= hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library initialization failed");
        return 1;
    }

    /* create and setup matrix descriptor */
    status= hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor initialization failed");
        return 1;
    }
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

    /* exercise conversion routines (convert matrix from COO 2 CSR format) */
    cudaStat1 = hipMalloc((void**)&csrRowPtr,(n+1)*sizeof(csrRowPtr[0]));
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Device malloc failed (csrRowPtr)");
        return 1;
    }
    status= hipsparseXcoo2csr(handle,cooRowIndex,nnz,n,
                             csrRowPtr,HIPSPARSE_INDEX_BASE_ZERO);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Conversion from COO to CSR format failed");
        return 1;
    }
    //csrRowPtr = [0 3 4 7 9]

    int devId;
    hipDeviceProp_t prop;
    hipError_t cudaStat;
    cudaStat = hipGetDevice(&devId);
    if (hipSuccess != cudaStat){
        CLEANUP("hipGetDevice failed");
        printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
        return 1;
    }
    cudaStat = hipGetDeviceProperties( &prop, devId) ;
    if (hipSuccess != cudaStat){
        CLEANUP("hipGetDeviceProperties failed");
        printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
        return 1;
    }

    /* SpmV */
    std::ofstream myfile;
    myfile.open ("example.txt");   
    printf("SpMV elapsed time:\n");
    for(int i = 0; i < 1000; i++){
        srand (time(NULL));
        for(int i = 0; i < n; i++){
            y_static[i] = rand() / double(RAND_MAX);
        }

        hipMemcpy(y, y_static, (size_t)(n*sizeof(y[0])), hipMemcpyHostToDevice);
        hipEventRecord(start);
        status= hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz,
                            &done, descr, cooVal, csrRowPtr, cooColIndex,
                            y, &dzero, y);
        hipEventRecord(stop);

        if (status != HIPSPARSE_STATUS_SUCCESS) {
            CLEANUP("Matrix-vector multiplication failed");
            return 1;
        }
        hipMemcpy(yHostPtr, y, (size_t)(n*sizeof(y[0])), hipMemcpyDeviceToHost);
        hipEventSynchronize(stop);
        float milliseconds = -1;
        hipEventElapsedTime(&milliseconds, start, stop); 
        myfile << 1000.0 * milliseconds << "\n";
        hipDeviceSynchronize();
    }
    myfile.close();

    /* destroy matrix descriptor */
    status = hipsparseDestroyMatDescr(descr);
    descr = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor destruction failed");
        return 1;
    }

    /* destroy handle */
    status = hipsparseDestroy(handle);
    handle = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library release of resources failed");
        return 1;
    }

    printf("SpMV results:\n");
    for (int j=0; j<1; j++){
        for (int i=0; i<n; i++){
            printf("yHostPtr[%d,%d]=%f\n",i,j,yHostPtr[i+n*j]);
        }
    }
    
    CLEANUP("example test PASSED");
    return 0;
}

