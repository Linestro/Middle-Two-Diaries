#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <fstream>
#include <time.h>

#define CLEANUP(s)                                   \
do {                                                 \
    printf ("%s\n", s);                              \
    if (yHostPtr)           free(yHostPtr);          \
    if (zHostPtr)           free(zHostPtr);          \
    if (xIndHostPtr)        free(xIndHostPtr);       \
    if (xValHostPtr)        free(xValHostPtr);       \
    if (cooRowIndexHostPtr) free(cooRowIndexHostPtr);\
    if (cooColIndexHostPtr) free(cooColIndexHostPtr);\
    if (cooValHostPtr)      free(cooValHostPtr);     \
    if (y)                  hipFree(y);             \
    if (z)                  hipFree(z);             \
    if (xInd)               hipFree(xInd);          \
    if (xVal)               hipFree(xVal);          \
    if (csrRowPtr)          hipFree(csrRowPtr);     \
    if (cooRowIndex)        hipFree(cooRowIndex);   \
    if (cooColIndex)        hipFree(cooColIndex);   \
    if (cooVal)             hipFree(cooVal);        \
    if (descr)              hipsparseDestroyMatDescr(descr);\
    if (handle)             hipsparseDestroy(handle); \
    hipDeviceReset();          \
    fflush (stdout);                                 \
} while (0)

int main(){
    hipError_t cudaStat1,cudaStat2,cudaStat3,cudaStat4;
    hipsparseStatus_t status;
    hipsparseHandle_t handle=0;
    hipsparseMatDescr_t descr=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int *    cooRowIndexHostPtr=0;
    int *    cooColIndexHostPtr=0;
    double * cooValHostPtr=0;
    int *    cooRowIndex=0;
    int *    cooColIndex=0;
    double * cooVal=0;
    int *    xIndHostPtr=0;
    double * xValHostPtr=0;
    double * yHostPtr=0;
    double * y_static=0;
    int *    xInd=0;
    double * xVal=0;
    double * y=0;
    int *    csrRowPtr=0;
    double * zHostPtr=0;
    double * z=0;
    int      n, nnz;
    double dzero =0.0;
    double done = 1.0;

    printf("testing example\n");
    /* create the following sparse test matrix in COO format */
    /* |1.0     2.0 3.0     |
       |    4.0             |
       |5.0     6.0 7.0     |
       |    8.0     9.0     |
       |                10.0| */

    n = 5;      // rank of the matrix
    nnz = 10;   // number of non-zero elements
    
    cooRowIndexHostPtr = (int *)   malloc(nnz*sizeof(cooRowIndexHostPtr[0]));
    cooColIndexHostPtr = (int *)   malloc(nnz*sizeof(cooColIndexHostPtr[0]));
    cooValHostPtr      = (double *)malloc(nnz*sizeof(cooValHostPtr[0]));
    if ((!cooRowIndexHostPtr) || (!cooColIndexHostPtr) || (!cooValHostPtr)){
        CLEANUP("Host malloc failed (matrix)");
        return 1;
    }
    cooRowIndexHostPtr[0]=0; cooColIndexHostPtr[0]=0; cooValHostPtr[0]=1.0;
    cooRowIndexHostPtr[1]=0; cooColIndexHostPtr[1]=2; cooValHostPtr[1]=2.0;
    cooRowIndexHostPtr[2]=0; cooColIndexHostPtr[2]=3; cooValHostPtr[2]=3.0;
    cooRowIndexHostPtr[3]=1; cooColIndexHostPtr[3]=1; cooValHostPtr[3]=4.0;
    cooRowIndexHostPtr[4]=2; cooColIndexHostPtr[4]=0; cooValHostPtr[4]=5.0;
    cooRowIndexHostPtr[5]=2; cooColIndexHostPtr[5]=2; cooValHostPtr[5]=6.0;
    cooRowIndexHostPtr[6]=2; cooColIndexHostPtr[6]=3; cooValHostPtr[6]=7.0;
    cooRowIndexHostPtr[7]=3; cooColIndexHostPtr[7]=1; cooValHostPtr[7]=8.0;
    cooRowIndexHostPtr[8]=3; cooColIndexHostPtr[8]=3; cooValHostPtr[8]=9.0;
    cooRowIndexHostPtr[9]=4; cooColIndexHostPtr[9]=4; cooValHostPtr[9]=10.0;
    /*
    //print the matrix
    printf("Input data:\n");
    for (int i=0; i<nnz; i++){
        printf("cooRowIndexHostPtr[%d]=%d  ",i,cooRowIndexHostPtr[i]);
        printf("cooColIndexHostPtr[%d]=%d  ",i,cooColIndexHostPtr[i]);
        printf("cooValHostPtr[%d]=%f     \n",i,cooValHostPtr[i]);
    }
    */

    /* create a dense vector */
    /*  y  = [1.0 2.0 3.0 4.0 5.0] (dense) */
    yHostPtr    = (double *)malloc(n       *sizeof(yHostPtr[0]));
    y_static    = (double *)malloc(n       *sizeof(yHostPtr[0]));
    if(!yHostPtr || !y_static){
        CLEANUP("Host malloc failed (vectors)");
        return 1;
    }

    srand (time(NULL));
    for(int i = 0; i < n; i++){
        y_static[i] = rand() / double(RAND_MAX);
    }

    /*
    //print the vectors
    for (int j=0; j<1; j++){
        for (int i=0; i<n; i++){
            printf("yHostPtr[%d,%d]=%f\n",i,j,yHostPtr[i+n*j]);
        }
    }
    */

    /* allocate GPU memory and copy the matrix and vectors into it */
    cudaStat1 = hipMalloc((void**)&cooRowIndex,nnz*sizeof(cooRowIndex[0]));
    cudaStat2 = hipMalloc((void**)&cooColIndex,nnz*sizeof(cooColIndex[0]));
    cudaStat3 = hipMalloc((void**)&cooVal,     nnz*sizeof(cooVal[0]));
    cudaStat4 = hipMalloc((void**)&y,          n*sizeof(y[0]));
    if ((cudaStat1 != hipSuccess) ||
        (cudaStat2 != hipSuccess) ||
        (cudaStat3 != hipSuccess) ||
        (cudaStat4 != hipSuccess)) {
        CLEANUP("Device malloc failed");
        return 1;
    }
    cudaStat1 = hipMemcpy(cooRowIndex, cooRowIndexHostPtr,
                           (size_t)(nnz*sizeof(cooRowIndex[0])),
                           hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(cooColIndex, cooColIndexHostPtr,
                           (size_t)(nnz*sizeof(cooColIndex[0])),
                           hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(cooVal,      cooValHostPtr,
                           (size_t)(nnz*sizeof(cooVal[0])),
                           hipMemcpyHostToDevice);
    cudaStat4 = hipMemcpy(y,           y_static,
                           (size_t)(n*sizeof(y[0])),
                           hipMemcpyHostToDevice);
    if ((cudaStat1 != hipSuccess) ||
        (cudaStat2 != hipSuccess) ||
        (cudaStat3 != hipSuccess) ||
        (cudaStat4 != hipSuccess)) {
        CLEANUP("Memcpy from Host to Device failed");
        return 1;
    }

    /* initialize cusparse library */
    status= hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library initialization failed");
        return 1;
    }

    /* create and setup matrix descriptor */
    status= hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor initialization failed");
        return 1;
    }
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

    /* exercise conversion routines (convert matrix from COO 2 CSR format) */
    cudaStat1 = hipMalloc((void**)&csrRowPtr,(n+1)*sizeof(csrRowPtr[0]));
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Device malloc failed (csrRowPtr)");
        return 1;
    }
    status= hipsparseXcoo2csr(handle,cooRowIndex,nnz,n,
                             csrRowPtr,HIPSPARSE_INDEX_BASE_ZERO);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Conversion from COO to CSR format failed");
        return 1;
    }
    //csrRowPtr = [0 3 4 7 9]

    int devId;
    hipDeviceProp_t prop;
    hipError_t cudaStat;
    cudaStat = hipGetDevice(&devId);
    if (hipSuccess != cudaStat){
        CLEANUP("hipGetDevice failed");
        printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
        return 1;
    }
    cudaStat = hipGetDeviceProperties( &prop, devId) ;
    if (hipSuccess != cudaStat){
        CLEANUP("hipGetDeviceProperties failed");
        printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
        return 1;
    }

    /* SpmV */
    std::ofstream myfile;
    myfile.open ("example.txt");   
    printf("SpMV elapsed time:\n");
    for(int i = 0; i < 100; i++){
        hipMemcpy(y, y_static, (size_t)(n*sizeof(y[0])), hipMemcpyHostToDevice);
        hipEventRecord(start);
        status= hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz,
                            &done, descr, cooVal, csrRowPtr, cooColIndex,
                            y, &dzero, y);
        hipEventRecord(stop);

        if (status != HIPSPARSE_STATUS_SUCCESS) {
            CLEANUP("Matrix-vector multiplication failed");
            return 1;
        }
        hipMemcpy(yHostPtr, y, (size_t)(n*sizeof(y[0])), hipMemcpyDeviceToHost);
        hipEventSynchronize(stop);
        float milliseconds = -1;
        hipEventElapsedTime(&milliseconds, start, stop); 
        myfile << 1000.0 * milliseconds << "\n";
        hipDeviceSynchronize();
    }
    myfile.close();

    /* destroy matrix descriptor */
    status = hipsparseDestroyMatDescr(descr);
    descr = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor destruction failed");
        return 1;
    }

    /* destroy handle */
    status = hipsparseDestroy(handle);
    handle = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library release of resources failed");
        return 1;
    }

    printf("SpMV results:\n");
    for (int j=0; j<1; j++){
        for (int i=0; i<n; i++){
            printf("yHostPtr[%d,%d]=%f\n",i,j,yHostPtr[i+n*j]);
        }
    }
    
    CLEANUP("example test PASSED");
    return 0;
}

